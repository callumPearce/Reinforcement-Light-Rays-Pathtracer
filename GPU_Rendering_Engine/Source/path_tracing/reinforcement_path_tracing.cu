#include "hip/hip_runtime.h"
#include "reinforcement_path_tracing.cuh"
//cuRand
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__
void update_radiance_volume_distributions(RadianceMap* radiance_map){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < radiance_map->radiance_volumes_count){
        radiance_map->radiance_volumes[i].update_radiance_distribution();
    }
}

__global__
void draw_reinforcement_path_tracing(vec3* device_buffer, hiprandState* d_rand_state, RadianceMap* radiance_map, Camera* camera, Scene* scene, int* device_path_lengths, int* zero_contribution_light_paths){
    
    // Populate the shared GPU/CPU screen buffer
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Path trace the ray to find the colour to paint the pixel
    device_buffer[x*(int)SCREEN_HEIGHT + y] = path_trace_reinforcement(d_rand_state, radiance_map, camera, x, y, scene, device_path_lengths, zero_contribution_light_paths);
}

__device__
vec3 path_trace_reinforcement(hiprandState* d_rand_state, RadianceMap* radiance_map, Camera* camera, int pixel_x, int pixel_y, Scene* scene, int* device_path_lengths, int* zero_contribution_light_paths){
    vec3 irradiance = vec3(0.f);
    int total_path_lengths = 0;
    for (int i = 0; i < SAMPLES_PER_PIXEL; i++){

        // Trace the path of the ray
        int path_length;
        vec3 temp_irradiance = path_trace_reinforcement_iterative(pixel_x, pixel_y, camera, d_rand_state, radiance_map, scene, path_length);
        irradiance += temp_irradiance;
        total_path_lengths += path_length;
        
        // Check if zero contribution light path
        float avg_temp_irradiance = (temp_irradiance.x + temp_irradiance.y + temp_irradiance.z)/3.f;
        if(avg_temp_irradiance < THROUGHPUT_THRESHOLD){
            atomicAdd(zero_contribution_light_paths, 1);
        }
    }
    int avg_path_length = int(total_path_lengths/SAMPLES_PER_PIXEL);
    device_path_lengths[pixel_x*SCREEN_HEIGHT + pixel_y] = avg_path_length;
    irradiance /= (float)SAMPLES_PER_PIXEL;
    return irradiance;
}

__device__
vec3 path_trace_reinforcement_iterative(int pixel_x, int pixel_y, Camera* camera, hiprandState* d_rand_state, RadianceMap* radiance_map, Scene* scene, int& path_length){

    Ray ray = Ray::sample_ray_through_pixel(d_rand_state, *camera, pixel_x, pixel_y);

    vec3 throughput = vec3(1.f);
    
    RadianceVolume* current_radiance_volume;
    int current_sector_x = -1;
    int current_sector_y = -1;
    float current_BRDF = 0.f;

    for (int i = 0; i < MAX_RAY_BOUNCES; i++){

        // Trace the path of the ray to find the closest intersection
        ray.closest_intersection(scene);

        // We cannot update Q on the first bounce as it is the camera position,
        // not a point in the scene. But we still need the closest radiance volume it intersects with
        if (i > 0){
            // Update Q
            // where x = ray.start, y = intersection.position
            // Check that a radiance volume has been found to update its sector
            if (current_radiance_volume && current_sector_x != -1 && current_sector_y != -1){
                current_radiance_volume = radiance_map->temporal_difference_update_radiance_volume_sector(current_BRDF, current_radiance_volume, current_sector_x, current_sector_y, ray.intersection, scene);
                current_sector_x = -1;
                current_sector_y = -1;
            } 
        }
        // Get the radiance volume for the first iteration
        else{
            if (ray.intersection.intersection_type == SURFACE)
                current_radiance_volume = radiance_map->find_closest_radiance_volume_iterative(MAX_DIST, ray.intersection.position, ray.intersection.normal);
        }

        // Check what they ray intersected with...
        switch(ray.intersection.intersection_type){
            // Interescted with nothing, so no radiance
            case NOTHING:
                path_length = i+1;
                return throughput * vec3(ENVIRONMENT_LIGHT);
                break;
            
            // Intersected with light plane, so return its diffuse_p
            case AREA_LIGHT:
                path_length= i+1;
                return throughput * scene->area_lights[ray.intersection.index].diffuse_p;
                break;

            // Intersected with a surface (diffuse)
            case SURFACE:

                vec4 sampled_direction = vec4(0.f);
                float pdf = 0.f;
                radiance_map->importance_sample_ray_direction(d_rand_state, ray.intersection, current_sector_x, current_sector_y, pixel_x, pixel_y, sampled_direction, current_radiance_volume, pdf);

                vec3 BRDF = scene->surfaces[ray.intersection.index].material.diffuse_c / (float)M_PI;
                float cos_theta = dot(vec3(scene->surfaces[ray.intersection.index].normal), vec3(sampled_direction));

                current_BRDF = (scene->surfaces[ray.intersection.index].material.luminance) / (float)M_PI;
                throughput *= (BRDF * cos_theta) / pdf;
                
                vec4 start = ray.intersection.position + sampled_direction * 0.00001f;
                start.w = 1.f;
                ray = Ray(start, sampled_direction);
                break;
            }
    }
    path_length = MAX_RAY_BOUNCES;
    return vec3(0);
}
