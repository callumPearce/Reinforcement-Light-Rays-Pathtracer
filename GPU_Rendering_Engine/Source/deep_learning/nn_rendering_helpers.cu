#include "hip/hip_runtime.h"
#include "nn_rendering_helpers.cuh"


// Randomly sample with the given grid index a 3D ray direction
__device__
void sample_ray_for_grid_index(
    hiprandState* d_rand_state,
    int grid_idx,
    float* ray_directions_device,
    float* ray_normals_device,
    float* ray_locations_device,
    float* ray_throughputs_device,
    int i
){
    // Convert the index to a grid position
    int dir_x = int(grid_idx/GRID_RESOLUTION);
    int dir_y = grid_idx - (dir_x*GRID_RESOLUTION);

    // Convert to 3D direction and update the direction
    vec3 position = vec3(ray_locations_device[(i*3)], ray_locations_device[(i*3) + 1], ray_locations_device[(i*3) + 2]);
    vec3 normal  = vec3(ray_normals_device[(i*3)], ray_normals_device[(i*3) + 1], ray_normals_device[(i*3) + 2]);
    mat4 transformation_matrix = create_transformation_matrix(normal, vec4(position, 1.f));
    vec3 dir = convert_grid_pos_to_direction_random(d_rand_state, (float) dir_x, (float) dir_y, i, position, transformation_matrix);
    ray_directions_device[(i*3)    ] = dir.x;
    ray_directions_device[(i*3) + 1] = dir.y;
    ray_directions_device[(i*3) + 2] = dir.z;

    // Update throughput with new sampled angle
    float cos_theta = dot(normal, dir);
    ray_throughputs_device[(i*3)    ] = (ray_throughputs_device[(i*3)    ] * cos_theta)/RHO;
    ray_throughputs_device[(i*3) + 1] = (ray_throughputs_device[(i*3) + 1] * cos_theta)/RHO;
    ray_throughputs_device[(i*3) + 2] = (ray_throughputs_device[(i*3) + 2] * cos_theta)/RHO;
}

// Randomly sample a ray within the given grid idx and return as vec3
__device__
vec3 sample_ray_for_grid_index(
    hiprandState* d_rand_state,
    int grid_idx,
    float* ray_normals_device,
    float* ray_locations_device,
    int i
){
    // Convert the index to a grid position
    int dir_x = int(grid_idx/GRID_RESOLUTION);
    int dir_y = grid_idx - (dir_x*GRID_RESOLUTION);

    // Convert to 3D direction and update the direction
    vec3 position = vec3(ray_locations_device[(i*3)], ray_locations_device[(i*3) + 1], ray_locations_device[(i*3) + 2]);
    vec3 normal  = vec3(ray_normals_device[(i*3)], ray_normals_device[(i*3) + 1], ray_normals_device[(i*3) + 2]);
    mat4 transformation_matrix = create_transformation_matrix(normal, vec4(position, 1.f));
    return convert_grid_pos_to_direction_random(d_rand_state, (float) dir_x, (float) dir_y, i, position, transformation_matrix);
}

// Sample random directions to further trace the rays in
__global__
void sample_next_ray_directions_randomly(
        hiprandState* d_rand_state,
        float* ray_normals, 
        float* ray_directions,
        float* ray_throughputs,
        bool* ray_terminated
){
    
    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;

    // Do nothing if we have already intersected with the light
    if (ray_terminated[i] == true){
        return;
    }

    // Sample the new direction and record it along with cos_theta
    float cos_theta;
    vec3 dir = vec3(sample_random_direction_around_intersection(d_rand_state, vec3(ray_normals[(i*3)], ray_normals[(i*3)+1], ray_normals[(i*3)+2]), cos_theta));
    ray_directions[(i*3)    ] = dir.x;
    ray_directions[(i*3) + 1] = dir.y;
    ray_directions[(i*3) + 2] = dir.z;

    // Update throughput with new sampled angle
    ray_throughputs[(i*3)    ] = (ray_throughputs[(i*3)    ] * cos_theta)/RHO;
    ray_throughputs[(i*3) + 1] = (ray_throughputs[(i*3) + 1] * cos_theta)/RHO;
    ray_throughputs[(i*3) + 2] = (ray_throughputs[(i*3) + 2] * cos_theta)/RHO;
}


// Compute the TD targets for the current batch size
__global__
void compute_td_targets(
    hiprandState* d_rand_state,
    float* next_qs_device,
    float* td_targets_device,
    float* ray_locations,
    float* ray_normals,
    float* ray_rewards,
    float* ray_discounts,
    int batch_start_idx
){
    int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
  
    if (batch_start_idx + batch_idx >= SCREEN_HEIGHT*SCREEN_WIDTH) return;

    // Get the max q_val
    unsigned int max_idx = 0;
    float max_q_val = next_qs_device[batch_idx];
    for (unsigned int i = 1; i < GRID_RESOLUTION*GRID_RESOLUTION; i++){
        float temp_q = next_qs_device[batch_idx + i];
        if (max_q_val < temp_q){
            max_q_val = temp_q;
            max_idx = i;
        }
    }

    // Calculate cos_theta
    vec3 dir = sample_ray_for_grid_index(
        d_rand_state,
        max_idx,
        ray_normals,
        ray_locations,
        (batch_idx+batch_start_idx)
    );
    vec3 normal(ray_normals[(batch_start_idx+batch_idx)*3], ray_normals[(batch_start_idx+batch_idx)*3 + 1], ray_normals[(batch_start_idx+batch_idx)*3 + 2]);
    float cos_theta = dot(normal, dir);

    //TODO: Fix this, cos_theta multiplication causing program crash
    td_targets_device[ batch_idx ] =  ray_rewards[ batch_idx + batch_start_idx ] + max_q_val*ray_discounts[ batch_idx + batch_start_idx ];//*cos_theta;
}   

// Update pixel values stored in the device_buffer
__global__
void update_total_throughput(
        float* ray_throughputs,
        vec3* total_throughputs
){

    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;
    
    // Accumulate
    total_throughputs[i] += vec3(ray_throughputs[(i*3)], ray_throughputs[(i*3)+1], ray_throughputs[(i*3)+2]);
}

// Update the device_buffer with the throughput
__global__
void update_device_buffer(
    vec3* device_buffer,
    vec3* total_throughputs
){
    
    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;

    // Update
    device_buffer[i] = total_throughputs[i]/(float)SAMPLES_PER_PIXEL;
}

// Sum up all path lengths
__global__
void sum_path_lengths(
    int* total_path_lengths_device,
    unsigned int* ray_bounces
){
    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;

    atomicAdd(total_path_lengths_device, (int)ray_bounces[i]); 
}

inline bool file_exists (const std::string& name) {
    struct stat buffer;   
    return (stat (name.c_str(), &buffer) == 0); 
}


// Read the scene data file and populate the list of vertices
void load_scene_data(Scene& scene, std::vector<float>& scene_data){

    for (int i = 0; i < scene.surfaces_count; i++){
        
        Surface sf = scene.surfaces[i];

        // Normals
        scene_data.push_back(sf.normal.x);
        scene_data.push_back(sf.normal.y);
        scene_data.push_back(sf.normal.z);

        // Vertices
        scene_data.push_back(sf.v0.x);
        scene_data.push_back(sf.v0.y);
        scene_data.push_back(sf.v0.z);
        scene_data.push_back(sf.v1.x);
        scene_data.push_back(sf.v1.y);
        scene_data.push_back(sf.v1.z);
        scene_data.push_back(sf.v2.x);
        scene_data.push_back(sf.v2.y);
        scene_data.push_back(sf.v2.z);
    }

    for (int i = 0; i < scene.area_light_count; i++){

        AreaLight al = scene.area_lights[i];

        // Normals
        scene_data.push_back(al.normal.x);
        scene_data.push_back(al.normal.y);
        scene_data.push_back(al.normal.z);

        // Vertices
        scene_data.push_back(al.v0.x);
        scene_data.push_back(al.v0.y);
        scene_data.push_back(al.v0.z);
        scene_data.push_back(al.v1.x);
        scene_data.push_back(al.v1.y);
        scene_data.push_back(al.v1.z);
        scene_data.push_back(al.v2.x);
        scene_data.push_back(al.v2.y);
        scene_data.push_back(al.v2.z);
    }
}