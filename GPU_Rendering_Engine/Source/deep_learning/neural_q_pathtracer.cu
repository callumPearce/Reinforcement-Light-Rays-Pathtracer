#include "hip/hip_runtime.h"
#include "neural_q_pathtracer.cuh"


__host__
NeuralQPathtracer::NeuralQPathtracer(
        unsigned int frames,
        unsigned int batch_size,
        SDLScreen& screen,
        Scene& scene,
        Camera& camera,
        int argc,
        char** argv
    ){

    //////////////////////////////////////////////////////////////
    /*                  Assign attributes                       */
    //////////////////////////////////////////////////////////////
    this->epsilon = EPSILON_START;
    this->vertices_count = scene.vertices_count;
    this->ray_batch_size = batch_size; /* How many rays to be processed at once */
    this->num_batches = (SCREEN_HEIGHT*SCREEN_WIDTH + (batch_size -1))/batch_size; /* How many batches in total */
    printf("Batch Size: %d\n", batch_size);
    printf("Number of Batches: %d\n", num_batches);

    dim3 b_size(8,8);
    this->block_size = b_size; /* How many threads in a single block to process the screen*/
    int blocks_x = (SCREEN_WIDTH + this->block_size.x - 1)/this->block_size.x;
    int blocks_y = (SCREEN_HEIGHT + this->block_size.y - 1)/this->block_size.y;
    dim3 n_bs(blocks_x, blocks_y);
    this->num_blocks = n_bs;/* How many blocks to process all pixels on the screen */

    //////////////////////////////////////////////////////////////
    /*                Initialise the DQN                        */
    //////////////////////////////////////////////////////////////
    //TODO: Might have to specify the amount of memory the GPU can use
    // beforehand, otherwise it seems to assign over memory allocated later
    // on. It may continue to do this when calculating back&forwad prop
    auto dyparams = dynet::extract_dynet_params(argc, argv);
    dynet::initialize(dyparams);
    dynet::ParameterCollection model;
    dynet::AdamTrainer trainer(model);
    this->dqn = DQNetwork();
    this->dqn.initialize(model, this->vertices_count + 3 /* Input dims */, GRID_RESOLUTION*GRID_RESOLUTION /* Output dims */);

    //////////////////////////////////////////////////////////////
    /*                Load the previous DQN Model               */
    //////////////////////////////////////////////////////////////
    std::string fname = "../Radiance_Map_Data/deep_q_learning.model";
    if (LOAD_MODEL && file_exists(fname)){
        dynet::TextFileLoader loader(fname);
        loader.populate(model);
    }

    //////////////////////////////////////////////////////////////
    /*          Intialise Pixel value buffers                   */
    //////////////////////////////////////////////////////////////
    vec3* host_buffer = new vec3[ SCREEN_HEIGHT * SCREEN_WIDTH ];
    vec3* device_buffer;
    checkCudaErrors(hipMalloc(&device_buffer, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(vec3)));

    //////////////////////////////////////////////////////////////
    /*          Initialise Prev Host buffers                    */
    //////////////////////////////////////////////////////////////
    float* prev_location_host = new float[ SCREEN_HEIGHT * SCREEN_WIDTH * 3 ];
    unsigned int* directions_host = new unsigned int[ SCREEN_HEIGHT * SCREEN_WIDTH ];
    float* host_vertices = new float[ scene.vertices_count ];
    memcpy(host_vertices, scene.vertices, sizeof(float) * this->vertices_count);
    
    //////////////////////////////////////////////////////////////
    /*          Initialise ray arrays on CUDA device            */
    //////////////////////////////////////////////////////////////
    float* ray_locations;   /* Ray intersection location (State) */
    float* ray_normals;     /* Intersection normal */
    float* ray_directions;  /* Direction to next shoot the ray */
    bool* ray_terminated;  /* Has the ray intersected with a light/nothing */
    float* ray_rewards;    /* Reward recieved from Q(s,a) */
    float* ray_discounts;  /* Discount factor for current rays path */
    float* ray_throughputs; /* Throughput for calc pixel value */
    unsigned int* ray_bounces; /* Total number of bounces for each ray before intersection*/

    checkCudaErrors(hipMalloc(&ray_locations, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_normals, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_directions, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_terminated, sizeof(bool) * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_rewards, sizeof(float) * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_discounts, sizeof(float) * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_throughputs, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_bounces, sizeof(unsigned int) *SCREEN_HEIGHT *SCREEN_WIDTH));
    
    Camera* device_camera; /* Camera on the CUDA device */
    Surface* device_surfaces;
    AreaLight* device_light_planes;
    float* device_vertices;
    Scene* device_scene;   /* Scene to render */

    // Copy the camera
    checkCudaErrors(hipMalloc(&device_camera, sizeof(Camera)));
    checkCudaErrors(hipMemcpy(device_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice));
    
    // Copy surfaces into device memory space
    checkCudaErrors(hipMalloc(&device_surfaces, scene.surfaces_count * sizeof(Surface)));
    checkCudaErrors(hipMemcpy(device_surfaces, scene.surfaces, scene.surfaces_count * sizeof(Surface), hipMemcpyHostToDevice));

    // Copy light planes into device memory space
    checkCudaErrors(hipMalloc(&device_light_planes, scene.area_light_count * sizeof(AreaLight)));
    checkCudaErrors(hipMemcpy(device_light_planes, scene.area_lights, scene.area_light_count * sizeof(AreaLight), hipMemcpyHostToDevice));    

    // Copy vertices into device memory space
    checkCudaErrors(hipMalloc(&device_vertices, scene.vertices_count * sizeof(float)));
    checkCudaErrors(hipMemcpy(device_vertices, scene.vertices, scene.vertices_count * sizeof(float), hipMemcpyHostToDevice));  

    // Copy the scene structure into the device and its corresponding pointers to Surfaces, Area Lights and Vertices
    checkCudaErrors(hipMalloc(&device_scene, sizeof(Scene)));
    checkCudaErrors(hipMemcpy(device_scene, &scene, sizeof(Scene), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&(device_scene->surfaces), &device_surfaces, sizeof(Surface*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&(device_scene->area_lights), &device_light_planes, sizeof(AreaLight*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&(device_scene->vertices), &device_vertices, sizeof(float*), hipMemcpyHostToDevice));

    //////////////////////////////////////////////////////////////
    /*                  Intialise cuRand State                  */
    //////////////////////////////////////////////////////////////
    hiprandState * d_rand_state;
    checkCudaErrors(hipMalloc(&d_rand_state, (float)SCREEN_HEIGHT * (float)SCREEN_WIDTH * sizeof(hiprandState)));
    init_rand_state<<<this->num_blocks, this->block_size>>>(d_rand_state, SCREEN_WIDTH, SCREEN_HEIGHT);

    //////////////////////////////////////////////////////////////
    /*                  Render the frames                       */
    //////////////////////////////////////////////////////////////
    for (int i = 0; i < frames; i++){
        //Clear the pixel buffer
        memset(host_buffer, 0.f, sizeof(vec3)* SCREEN_HEIGHT * SCREEN_WIDTH);

        /* Compute frame time */
        static int t = SDL_GetTicks();
        int t2 = SDL_GetTicks();
        float dt = float(t2-t);
        t = t2;
        printf("Render Time: %.3f ms.\n", dt);
        
        // Fill the pixel buffer each frame using Deep Q-Learning strategy
        this->render_frame(
            trainer, 
            d_rand_state,
            host_vertices,
            device_camera,
            device_scene,
            device_buffer,
            prev_location_host,
            directions_host,
            ray_locations,
            ray_normals,   
            ray_directions,
            ray_terminated,  
            ray_rewards,   
            ray_discounts, 
            ray_throughputs,
            ray_bounces
        );

        // Copy the device buffer values to the host buffer
        checkCudaErrors(hipMemcpy(host_buffer, device_buffer, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(vec3), hipMemcpyDeviceToHost));

        // Display the rendered frame
        for (int x = 0; x < SCREEN_WIDTH; x++){
            for (int y = 0; y < SCREEN_HEIGHT; y++){
                screen.PutPixelSDL(x, y, host_buffer[x*(int)SCREEN_HEIGHT + y]);
            }
        }
        screen.SDL_Renderframe();

        //////////////////////////////////////////////////////////////
        /*                  Save the DQN Model                      */
        //////////////////////////////////////////////////////////////
        std::cout << "Saving model..." << std::endl;
        if (SAVE_MODEL){
            dynet::TextFileSaver saver(fname);
            saver.save(model);
        }
        std::cout << "Model saved." << std::endl;
    }

    //////////////////////////////////////////////////////////////
    /*          Save the image and kill the screen              */
    //////////////////////////////////////////////////////////////
    screen.SDL_SaveImage("../Images/render.bmp");
    screen.kill_screen();

    //////////////////////////////////////////////////////////////
    /*                      Free memory used                    */
    //////////////////////////////////////////////////////////////
    delete [] host_buffer;
    delete [] prev_location_host;
    delete [] directions_host;
    delete [] host_vertices;
    hipFree(device_buffer);
    hipFree(d_rand_state);
    hipFree(ray_locations);
    hipFree(ray_normals);
    hipFree(ray_directions);
    hipFree(ray_terminated);
    hipFree(ray_rewards);
    hipFree(ray_throughputs);
    hipFree(ray_bounces);
    hipFree(device_camera);
    hipFree(device_surfaces);
    hipFree(device_light_planes);
    hipFree(device_vertices);
    hipFree(device_scene);
}

__host__
void NeuralQPathtracer::render_frame(
        dynet::AdamTrainer trainer,
        hiprandState* d_rand_state,
        float* host_vertices,
        Camera* device_camera,
        Scene* device_scene,
        vec3* device_buffer,
        float* prev_location_host,
        unsigned int* directions_host,
        float* ray_locations,   /* Ray intersection location (State) */
        float* ray_normals,     /* Intersection normal */
        float* ray_directions,  /* Direction to next shoot the ray */
        bool* ray_terminated,  /* Has the ray intersected with a light/nothing */
        float* ray_rewards,    /* Reward recieved from Q(s,a) */
        float* ray_discounts,  /* Discount factor for current rays path */
        float* ray_throughputs,  /* Throughput for calc pixel value */
        unsigned int* ray_bounces /* Total number of bounces for each ray before intersection*/
    ){

    // Initialise buffer to hold total throughput
    vec3* total_throughputs;
    checkCudaErrors(hipMalloc(&total_throughputs, sizeof(vec3) * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMemset(total_throughputs, 0.f, sizeof(vec3) * SCREEN_HEIGHT * SCREEN_WIDTH));
    
    // Sample through each pixel SAMPLES_PER_PIXEL times
    for (int i = 0; i < SAMPLES_PER_PIXEL; i++){
        // Initialise ray variables
        initialise_ray<<<this->num_blocks, this->block_size>>>(
            d_rand_state,
            device_camera, 
            ray_locations, 
            ray_directions,
            ray_terminated, 
            ray_rewards, 
            ray_discounts,
            ray_throughputs,
            ray_bounces
        );
        checkCudaErrors(hipDeviceSynchronize());

        // Create bool to determine if all rays in the batch have collided with a light
        int rays_finished = 0; /* If not updated to false by trace_ray, end loop */
        int* device_rays_finished;
        checkCudaErrors(hipMalloc(&device_rays_finished, sizeof(int)));
        checkCudaErrors(hipMemset(device_rays_finished, 1, sizeof(int)));

        // Trace rays path until all have intersected with a light
        unsigned int bounces = 0;
        float loss = 0.f;
        while(rays_finished == 0 && bounces < MAX_RAY_BOUNCES){

            printf("Bounce: %d/%d\n", bounces, MAX_RAY_BOUNCES);

            // Maintain previous locations for reinforcment Q(s,a) update
            checkCudaErrors(hipMemcpy(prev_location_host, ray_locations, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH, hipMemcpyDeviceToHost));

            // Does not apply to shooting from camera
            if (bounces > 0){

                // For each batch sample Q-values and apply eta-greedy policy
                for(int n = 0; n < this->num_batches; n++){
                    
                    // Compute Batch-Size
                    unsigned int current_batch_size = std::min(SCREEN_HEIGHT*SCREEN_WIDTH - (n*this->ray_batch_size), this->ray_batch_size);

                    // Initialise the graph
                    dynet::ComputationGraph graph;

                    // Formulate the input expression
                    // dynet::Dim input_dim({3},current_batch_size);
                    // std::vector<float> input_states(3*current_batch_size);
                    // memcpy(&(input_states[0]), &(prev_location_host[n*this->ray_batch_size*3]), sizeof(float) * 3 * current_batch_size);
                    // dynet::Expression states_batch = dynet::input(graph, input_dim, input_states); //TODO might need pointer on last element

                    // Formulate the expression with the state and the scenes vertices
                    dynet::Dim input_dim({3 + (unsigned int)this->vertices_count},current_batch_size);
                    std::vector<float> input_vals(3*current_batch_size + this->vertices_count*current_batch_size);
                    for (int s = 0; s < current_batch_size; s++){
                        // Copy the current position
                        memcpy(
                            &(input_vals[3*s + this->vertices_count*s]),             // dst
                            &(prev_location_host[n*this->ray_batch_size*3 + 3*s]), // src 
                            sizeof(float) * 3                                      // count
                        );
                        // Copy the list off all vertices
                        memcpy(
                            &(input_vals[3*s + this->vertices_count*s + 3]),
                            host_vertices,
                            sizeof(float) * this->vertices_count
                        );
                    }
                    dynet::Expression states_batch = dynet::input(graph, input_dim, input_vals); 
                    
                    // Get the Q-values
                    dynet::Expression prediction = this->dqn.network_inference(graph, states_batch, false);
                    std::vector<float> current_qs = dynet::as_vector(graph.forward(prediction));

                    // Copy Q-vals to GPU for find the argmax
                    float* current_qs_device;
                    checkCudaErrors(hipMalloc(&current_qs_device, sizeof(float) * current_qs.size()));
                    checkCudaErrors(hipMemcpy(current_qs_device, &(current_qs[0]), sizeof(float) * current_qs.size() , hipMemcpyHostToDevice));

                    // Setup the deivce storage for the ray direction indices
                    unsigned int* ray_direction_indices;
                    checkCudaErrors(hipMalloc(&ray_direction_indices, sizeof(unsigned int) * current_batch_size));

                    // Get direction indices (Call once for every element in the batch)
                    int threads = 32;
                    int blocks = (current_batch_size + (threads-1))/threads;
                    sample_batch_ray_directions_eta_greedy<<<blocks, threads>>>(
                        this->epsilon,
                        d_rand_state,
                        ray_direction_indices,
                        current_qs_device,
                        ray_directions,
                        ray_locations,
                        ray_normals,
                        ray_throughputs,
                        ray_terminated,
                        (n*this->ray_batch_size)
                    );
                    hipDeviceSynchronize();

                    // Copy over ray_direction_indices to the host
                    checkCudaErrors(hipMemcpy(&(directions_host[ this->ray_batch_size * n ]), ray_direction_indices, sizeof(unsigned int) * current_batch_size, hipMemcpyDeviceToHost));

                    // Free memory
                    hipFree(ray_direction_indices);
                    hipFree(current_qs_device);
                }
            }

            // Trace the rays in their set directions
            trace_ray<<<this->num_blocks, this->block_size>>>(
                device_scene,
                d_rand_state,
                device_rays_finished,
                ray_locations, 
                ray_normals,
                ray_directions, 
                ray_terminated, 
                ray_rewards,
                ray_discounts,
                ray_throughputs,
                ray_bounces,
                bounces
            );  
            hipDeviceSynchronize();

            // Does not apply to shooting from camera //TODO: We are current backpropogating rays that have terminated continually, this is bad
            if(bounces > 0){

                // Copy data from Cuda device to host for usage
                float* ray_locations_host = new float[ SCREEN_HEIGHT * SCREEN_WIDTH * 3 ];
                checkCudaErrors(hipMemcpy(ray_locations_host, ray_locations, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH , hipMemcpyDeviceToHost));
                // Run learning rule on the network with the results received and sample new direction for each ray in batches
                for(int n = 0; n < this->num_batches; n++){
                    
                    dynet::ComputationGraph graph;

                    // 1) Create the input expression to the neural network for S_t+1
                    unsigned int current_batch_size = std::min(SCREEN_HEIGHT*SCREEN_WIDTH - (n*this->ray_batch_size), this->ray_batch_size);

                    // Get the input batch for the new ray positions
                    // dynet::Dim input_dim({3},current_batch_size);
                    // std::vector<float> input_vals(3*current_batch_size);
                    // memcpy(&(input_vals[0]), &(ray_locations_host[n*current_batch_size*3]), sizeof(float) * 3 * current_batch_size);
                    // dynet::Expression input_batch = dynet::input(graph, input_dim, &input_vals);

                    // Formulate the expression with the state and the scenes vertices
                    dynet::Dim input_dim({3 + (unsigned int)this->vertices_count},current_batch_size);
                    std::vector<float> input_vals(3*current_batch_size + this->vertices_count*current_batch_size);
                    for (int s = 0; s < current_batch_size; s++){
                        // Copy the current position
                        memcpy(
                            &(input_vals[3*s + this->vertices_count*s]),             // dst
                            &(ray_locations_host[n*this->ray_batch_size*3 + 3*s]), // src 
                            sizeof(float) * 3                                      // count
                        );
                        // Copy the list off all vertices
                        memcpy(
                            &(input_vals[3*s + this->vertices_count*s + 3]),
                            host_vertices,
                            sizeof(float) * this->vertices_count
                        );
                    }
                    dynet::Expression input_batch = dynet::input(graph, input_dim, input_vals); 

                    // 2) Get max_a Q(S_{t+1}, a)
                    dynet::Expression next_qs_expr = this->dqn.network_inference(graph, input_batch, false);
                    std::vector<float> next_qs = dynet::as_vector(graph.forward(next_qs_expr));

                    // 3) Compute TD-Targets
                    float* next_qs_device;
                    checkCudaErrors(hipMalloc(&next_qs_device, sizeof(float) * current_batch_size * GRID_RESOLUTION * GRID_RESOLUTION));
                    checkCudaErrors(hipMemcpy(next_qs_device, &(next_qs[0]), sizeof(float) * current_batch_size * GRID_RESOLUTION * GRID_RESOLUTION, hipMemcpyHostToDevice));

                    float* td_targets_device;
                    checkCudaErrors(hipMalloc(&td_targets_device, sizeof(float) * current_batch_size));

                    int threads = 32;
                    int blocks = int((current_batch_size + (threads-1))/threads);
                    compute_td_targets<<<blocks, threads>>>(
                        d_rand_state,
                        next_qs_device,
                        td_targets_device,
                        ray_locations,
                        ray_normals,
                        ray_rewards,
                        ray_discounts,
                        (n*this->ray_batch_size)
                    );
                    hipDeviceSynchronize();

                    hipFree(next_qs_device);

                    std::vector<float> td_targets(current_batch_size);
                    checkCudaErrors(hipMemcpy(&(td_targets[0]), td_targets_device, sizeof(float) * current_batch_size, hipMemcpyDeviceToHost));
                    hipFree(td_targets_device);

                    // 4) Reset computational graph and use target_value as a constant
                    graph.clear();
                    dynet::Expression td_target = dynet::input(graph, dynet::Dim({1}, current_batch_size), td_targets);

                    // // 5) Get current Q(s,a) value
                    // std::vector<float> input_states(3*current_batch_size);
                    // memcpy(&(input_states[0]), &prev_location_host[n*current_batch_size*3], sizeof(float) * 3 * current_batch_size);
                    // dynet::Expression states_batch = dynet::input(graph, input_dim, input_states);
                    
                    // Formulate the expression with the state and the scenes vertices
                    std::vector<float> input_states(3*current_batch_size + this->vertices_count*current_batch_size);
                    for (int s = 0; s < current_batch_size; s++){
                        // Copy the current position
                        memcpy(
                            &(input_states[3*s + this->vertices_count*s]),         // dst
                            &(prev_location_host[n*this->ray_batch_size*3 + 3*s]), // src 
                            sizeof(float) * 3                                      // count
                        );
                        // Copy the list off all vertices
                        memcpy(
                            &(input_states[3*s + this->vertices_count*s + 3]),
                            host_vertices,
                            sizeof(float) * this->vertices_count
                        );
                    }
                    dynet::Expression states_batch = dynet::input(graph, input_dim, input_states); 

                    dynet::Expression prediction_qs = this->dqn.network_inference(graph, states_batch, true);
                    
                    // Get the vector of action value indices we took 
                    std::vector<unsigned int> action_value_indices(current_batch_size);
                    memcpy(&(action_value_indices[0]), &directions_host[this->ray_batch_size*n], sizeof(unsigned int) * current_batch_size);

                    // Get the current Q values for the actions taken
                    dynet::Expression current_qs = dynet::pick(prediction_qs, action_value_indices, (unsigned int) 0);
                    
                    // 6) Calculate the loss
                    dynet::Expression loss_expr = dynet::pow((td_target - current_qs), dynet::input(graph, 2.f));  
                    loss_expr = dynet::sum_batches(loss_expr); 
                    loss += dynet::as_scalar(graph.forward(loss_expr));

                    // 7) Train the network
                    graph.backward(loss_expr);
                    trainer.update();
                }

                // Dete the host arrays
                delete [] ray_locations_host;
            }

            // Copy over value to check if all rays have intersected with a light
            checkCudaErrors(hipMemcpy(&rays_finished, device_rays_finished, sizeof(int), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemset(device_rays_finished, 1, sizeof(int)));

            // Increment the number of bounces
            bounces++;
        }

        // Update epsilon
        this->epsilon = std::max( this->epsilon - EPSILON_DECAY, EPSILON_MIN);

        printf("epsilon: %.3f\n", this->epsilon);

        printf("loss: %.3f\n",loss);

        // Add computed throughput values to the running total
        update_total_throughput<<<this->num_blocks, this->block_size>>>(
            ray_throughputs,
            total_throughputs
        );
        hipDeviceSynchronize();
        hipFree(device_rays_finished);
    }
    // Update the device_buffer with the throughput
    update_device_buffer<<<this->num_blocks, this->block_size>>>(
        device_buffer,
        total_throughputs
    );
    hipDeviceSynchronize();
    hipFree(total_throughputs);

    // Calculate the average path length
    int* total_path_lengths_device;
    checkCudaErrors(hipMalloc(&total_path_lengths_device, sizeof(int)));
    checkCudaErrors(hipMemset(total_path_lengths_device, 0, sizeof(int)));
    sum_path_lengths<<<this->num_blocks, this->block_size>>>(
        total_path_lengths_device,
        ray_bounces
    );
    int total_path_lengths = 0;
    checkCudaErrors(hipMemcpy(&total_path_lengths, total_path_lengths_device, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "Avg Path Length: " << total_path_lengths/(SCREEN_HEIGHT*SCREEN_WIDTH) << std::endl;
}

// Gets the initial direction to shoot a ray in
__global__
void initialise_ray(
        hiprandState* d_rand_state,
        Camera* device_camera, 
        float* ray_locations, 
        float* ray_directions,
        bool* ray_terminated, 
        float* ray_rewards, 
        float* ray_discounts,
        float* ray_throughputs,
        unsigned int* ray_bounces
    ){

    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;

    // Randomly sample a ray within the pixel
    Ray r = Ray::sample_ray_through_pixel(d_rand_state, *device_camera, x, y);
    ray_locations[(i*3)    ] = r.start.x;
    ray_locations[(i*3) + 1] = r.start.y;
    ray_locations[(i*3) + 2] = r.start.z;
    ray_directions[(i*3)    ] = r.direction.x;
    ray_directions[(i*3) + 1] = r.direction.y;
    ray_directions[(i*3) + 2] = r.direction.z;

    // Initialise ray_variables
    ray_rewards[i] = 0.f;
    ray_terminated[i] = false;
    ray_throughputs[(i*3)    ] = 1.f;
    ray_throughputs[(i*3) + 1] = 1.f;
    ray_throughputs[(i*3) + 2] = 1.f;
    ray_discounts[i] = 1.f;
    ray_bounces[i] = MAX_RAY_BOUNCES;
}

// Trace a ray for all ray locations given in the angles specified within the scene
__global__
void trace_ray(
        Scene* scene,
        hiprandState* d_rand_state,
        int* rays_finished,
        float* ray_locations, 
        float* ray_normals, 
        float* ray_directions,
        bool* ray_terminated, 
        float* ray_rewards,
        float* ray_discounts, 
        float* ray_throughputs,
        unsigned int* ray_bounces,
        int bounces
    ){
    
    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;


    // For the current ray, get its next state by shooting a ray in the direction stored in ray_directions
    vec3 position = vec3(ray_locations[(i*3)], ray_locations[(i*3)+1], ray_locations[(i*3)+2]);
    vec3 dir = vec3(ray_directions[(i*3)], ray_directions[(i*3)+1], ray_directions[(i*3)+2]);

    // Create the ray and trace it
    Ray ray(vec4(position + (dir * 0.00001f), 1.f), vec4(dir, 1.f));
    ray.closest_intersection(scene);

    // Update position, normal, and discount factor based on intersection
    switch(ray.intersection.intersection_type){

        // TERMINAL STATE: R_(t+1) = Environment light power
        case NOTHING:
            ray_rewards[i] = 0.f;
            ray_discounts[i] = 0.f;

            if ( !ray_terminated[i] ){
                ray_throughputs[(i*3)] = ray_throughputs[(i*3)] * ENVIRONMENT_LIGHT;
                ray_throughputs[(i*3)+1] = ray_throughputs[(i*3)+1] * ENVIRONMENT_LIGHT;
                ray_throughputs[(i*3)+2] = ray_throughputs[(i*3)+2] * ENVIRONMENT_LIGHT;
                ray_terminated[i] = true;
                ray_bounces[i] = (unsigned int)bounces;
            }

            // Sample a random starting position for ray on any surface in the scene to continue training
            sample_random_scene_pos(
                scene,
                d_rand_state,
                ray_normals,
                ray_locations,
                i
            );
            break;
        
        // TERMINAL STATE: R_(t+1) = Area light power
        case AREA_LIGHT:
            float diffuse_light_power = scene->area_lights[ray.intersection.index].luminance; 
            ray_rewards[i] = diffuse_light_power;
            ray_discounts[i] = 0.f;

            if ( !ray_terminated[i] ){
                vec3 diffuse_p = scene->area_lights[ray.intersection.index].diffuse_p;
                ray_throughputs[(i*3)] = ray_throughputs[(i*3)] * diffuse_p.x;
                ray_throughputs[(i*3)+1] = ray_throughputs[(i*3)+1] * diffuse_p.y;
                ray_throughputs[(i*3)+2] = ray_throughputs[(i*3)+2] * diffuse_p.z;
                ray_terminated[i] = true;
                ray_bounces[i] = (unsigned int)bounces;
            }

            // Sample a random starting position for ray on any surface in the scene to continue training
            sample_random_scene_pos(
                scene,
                d_rand_state,
                ray_normals,
                ray_locations,
                i
            );
            break;

        // NON-TERMINAL STATE: R_(t+1) + \gamma * max_a Q(S_t+1, a) 
        // where  R_(t+1) = 0 for diffuse surfaces
        case SURFACE:
            vec3 new_loc = vec3(ray.intersection.position);
            ray_locations[(i*3)  ] = new_loc.x;
            ray_locations[(i*3)+1] = new_loc.y;
            ray_locations[(i*3)+2] = new_loc.z;

            vec3 new_norm = ray.intersection.normal;
            ray_normals[(i*3)  ] = new_norm.x; 
            ray_normals[(i*3)+1] = new_norm.y;
            ray_normals[(i*3)+2] = new_norm.z;

            vec3 BRDF = scene->surfaces[ray.intersection.index].material.diffuse_c;
            
            // Get luminance of material
            float max_rgb = max(BRDF.x, BRDF.y);
            max_rgb = max(BRDF.z, max_rgb);
            float min_rgb = min(BRDF.x, BRDF.y);
            min_rgb = min(BRDF.z, min_rgb);
            float luminance = 0.5f * (max_rgb + min_rgb);

            // discount_factors holds cos_theta currently, update rgb throughput first
            if ( !ray_terminated[i] ){
                ray_throughputs[(i*3)] = ray_throughputs[(i*3)] * (BRDF.x / (float)M_PI);
                ray_throughputs[(i*3)+1] = ray_throughputs[(i*3)+1] * (BRDF.y / (float)M_PI);
                ray_throughputs[(i*3)+2] = ray_throughputs[(i*3)+2] * (BRDF.z / (float)M_PI);
            }

            // Now update discount_factors with luminance
            ray_rewards[i] = 0.f;
            ray_discounts[i] = luminance;

            // Still a ray being to bounce, so not finished
            if ( !ray_terminated[i] ){
                atomicExch(rays_finished, 0);
            }
            break;
    }
}

// Sample index directions according the neural network q vals
__global__
void sample_batch_ray_directions_eta_greedy(
    float eta,
    hiprandState* d_rand_state,
    unsigned int* ray_direction_indices,
    float* current_qs_device,
    float* ray_directions,
    float* ray_locations,
    float* ray_normals,
    float* ray_throughputs,
    bool* ray_terminated,
    int batch_start_idx
){
    // Get the index of the ray in the current batch
    int batch_elem =  blockIdx.x * blockDim.x + threadIdx.x;
    
    // Sample the random number to be used for eta-greedy policy
    float rv = hiprand_uniform(&d_rand_state[ batch_start_idx + batch_elem ]);

    // The total number of actions to choose from
    int action_count = GRID_RESOLUTION*GRID_RESOLUTION;

    // Direction grid idx to convert to 3D direction
    unsigned int direction_grid_idx = 0;

    // Greedy
    if (rv > eta){
        // Get the larget q-values index
        unsigned int max_idx = 0;
        float max_q = current_qs_device[ action_count * batch_elem ];
        for (unsigned int n = 0; n < action_count; n++){
            if (current_qs_device[ action_count * batch_elem + n] > max_q){
                max_idx = n;
                max_q = current_qs_device[ action_count * batch_elem + n];
            }
        }
        direction_grid_idx = max_idx;
    }
    // Explore
    else{
        // Sample a random grid index
        direction_grid_idx = 
            (unsigned int)((hiprand_uniform(&d_rand_state[ batch_start_idx + batch_elem ]) - 0.0001f) * action_count);
    }

    // Update the direction index storage
    ray_direction_indices[ batch_elem ] = direction_grid_idx;

    // // Convert the found grid idx to a 3D direction and store in ray_directions
    sample_ray_for_grid_index(
        d_rand_state,
        (int)direction_grid_idx,
        ray_directions,
        ray_normals,
        ray_locations,
        ray_throughputs,
        ray_terminated,
        (batch_start_idx + batch_elem)
    );
}
