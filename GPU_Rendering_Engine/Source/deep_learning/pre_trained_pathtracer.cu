#include "hip/hip_runtime.h"
#include "pre_trained_pathtracer.cuh"

inline bool file_exists (const std::string& name) {
    struct stat buffer;   
    return (stat (name.c_str(), &buffer) == 0); 
}

// Constructor
__host__
PretrainedPathtracer::PretrainedPathtracer(
    unsigned int frames,
    int batch_size, 
    SDLScreen& screen, 
    Scene& scene,
    Camera& camera,
    int argc,
    char** argv
){

    //////////////////////////////////////////////////////////////
    /*                  Assign attributes                       */
    //////////////////////////////////////////////////////////////
    this->batch_size = batch_size;
    this->num_batches = (SCREEN_HEIGHT*SCREEN_WIDTH + (batch_size -1))/batch_size;
    dim3 b_size(8,8);
    this->block_size = b_size;
    int blocks_x = (SCREEN_WIDTH + this->block_size.x - 1)/this->block_size.x;
    int blocks_y = (SCREEN_HEIGHT + this->block_size.y - 1)/this->block_size.y;
    dim3 n_bs(blocks_x, blocks_y);
    this->num_blocks = n_bs;

    //////////////////////////////////////////////////////////////
    /*                Initialise the DQN                        */
    //////////////////////////////////////////////////////////////
    auto dyparams = dynet::extract_dynet_params(argc, argv);
    dynet::initialize(dyparams);
    dynet::ParameterCollection model;
    this->dqn = DQNetwork();
    this->dqn.initialize(model, GRID_RESOLUTION*GRID_RESOLUTION);

    //////////////////////////////////////////////////////////////
    /*             Load in the Parameter Values                 */
    //////////////////////////////////////////////////////////////
    std::string fname = "/home/calst/Documents/year4/thesis/monte_carlo_raytracer/Radiance_Map_Data/radiance_map_model.model";
    if (file_exists(fname)){
        dynet::TextFileLoader loader(fname);
        loader.populate(model);
    }
    else{
        printf("Failed to load model, terminating program.\n");
        return;
    }

    //////////////////////////////////////////////////////////////
    /*          Intialise Pixel value buffers                   */
    //////////////////////////////////////////////////////////////
    vec3* host_buffer = new vec3[ SCREEN_HEIGHT * SCREEN_WIDTH ];
    vec3* device_buffer;
    checkCudaErrors(hipMalloc(&device_buffer, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(vec3)));

    //////////////////////////////////////////////////////////////
    /*               Initialise device buffers                  */
    //////////////////////////////////////////////////////////////
    float* ray_directions_device;   /* Direction to next shoot the ray (3D) */
    float* ray_locations_device;    /* Current intersection location of the ray (3D) */
    float* ray_normals_device;      /* Current intersected surfaces normal for the ray (3D) */
    bool* ray_terminated_device;    /* Has the ray intersected with a light/nothing? */
    float* ray_throughputs_device;  /* RGB scalars representing current colour throughput of the ray (3D) */
    unsigned int* ray_bounces_device;      /* Number of time the ray has bounced */

    checkCudaErrors(hipMalloc(&ray_directions_device, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_locations_device, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_normals_device, sizeof(float) * 3 *SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_terminated_device, sizeof(bool) * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_throughputs_device, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_bounces_device, sizeof(unsigned int) * SCREEN_HEIGHT * SCREEN_WIDTH));

    Camera* device_camera; /* Camera on the CUDA device */
    Surface* device_surfaces;
    AreaLight* device_light_planes;
    Scene* device_scene;   /* Scene to render */

    // Copy the camera
    checkCudaErrors(hipMalloc(&device_camera, sizeof(Camera)));
    checkCudaErrors(hipMemcpy(device_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice));
    
    // Copy surfaces into device memory space
    checkCudaErrors(hipMalloc(&device_surfaces, scene.surfaces_count * sizeof(Surface)));
    checkCudaErrors(hipMemcpy(device_surfaces, scene.surfaces, scene.surfaces_count * sizeof(Surface), hipMemcpyHostToDevice));

    // Copy light planes into device memory space
    checkCudaErrors(hipMalloc(&device_light_planes, scene.area_light_count * sizeof(AreaLight)));
    checkCudaErrors(hipMemcpy(device_light_planes, scene.area_lights, scene.area_light_count * sizeof(AreaLight), hipMemcpyHostToDevice));    

    // Copy the scene structure into the device and its corresponding pointers to Surfaces and Area Lights
    checkCudaErrors(hipMalloc(&device_scene, sizeof(Scene)));
    checkCudaErrors(hipMemcpy(device_scene, &scene, sizeof(Scene), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&(device_scene->surfaces), &device_surfaces, sizeof(Surface*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&(device_scene->area_lights), &device_light_planes, sizeof(AreaLight*), hipMemcpyHostToDevice));    

    //////////////////////////////////////////////////////////////
    /*                  Intialise cuRand State                  */
    //////////////////////////////////////////////////////////////
    hiprandState * d_rand_state;
    checkCudaErrors(hipMalloc(&d_rand_state, (float)SCREEN_HEIGHT * (float)SCREEN_WIDTH * sizeof(hiprandState)));
    init_rand_state<<<this->num_blocks, this->block_size>>>(d_rand_state, SCREEN_WIDTH, SCREEN_HEIGHT);

    //////////////////////////////////////////////////////////////
    /*                  Render the frames                       */
    //////////////////////////////////////////////////////////////
    for (int i = 0; i < frames; i++){
        //Clear the pixel buffer
        memset(host_buffer, 0.f, sizeof(vec3)* SCREEN_HEIGHT * SCREEN_WIDTH);

        /* Compute frame time */
        static int t = SDL_GetTicks();
        int t2 = SDL_GetTicks();
        float dt = float(t2-t);
        t = t2;
        printf("Render Time: %.3f ms.\n", dt);
        
        // Fill the pixel buffer each frame using Deep Q-Learning strategy
        this->render_frame(
            d_rand_state,
            device_camera,
            device_scene,
            device_buffer,
            ray_locations_device,
            ray_normals_device,   
            ray_directions_device,
            ray_terminated_device,  
            ray_throughputs_device,
            ray_bounces_device
        );

        // Copy the device buffer values to the host buffer
        checkCudaErrors(hipMemcpy(host_buffer, device_buffer, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(vec3), hipMemcpyDeviceToHost));

        // Display the rendered frame
        for (int x = 0; x < SCREEN_WIDTH; x++){
            for (int y = 0; y < SCREEN_HEIGHT; y++){
                screen.PutPixelSDL(x, y, host_buffer[x*(int)SCREEN_HEIGHT + y]);
            }
        }
        screen.SDL_Renderframe();
    }

    //////////////////////////////////////////////////////////////
    /*          Save the image and kill the screen              */
    //////////////////////////////////////////////////////////////
    screen.SDL_SaveImage("/home/calst/Documents/year4/thesis/monte_carlo_raytracer/Images/render.bmp");
    screen.kill_screen();

    //////////////////////////////////////////////////////////////
    /*                      Free memory used                    */
    //////////////////////////////////////////////////////////////
    delete [] host_buffer;
    hipFree(device_buffer);
    hipFree(d_rand_state);
    hipFree(ray_locations_device);
    hipFree(ray_normals_device);
    hipFree(ray_directions_device);
    hipFree(ray_terminated_device);
    hipFree(ray_throughputs_device);
    hipFree(ray_bounces_device);
}

// Render a frame to output
__host__
void PretrainedPathtracer::render_frame(
    hiprandState* d_rand_state,
    Camera* device_camera,
    Scene* device_scene,
    vec3* device_buffer,
    float* ray_locations_device,
    float* ray_normals_device,   
    float* ray_directions_device,
    bool* ray_terminated_device,  
    float* ray_throughputs_device,
    unsigned int* ray_bounces_device
){
    // Initialise the buffer to hold the total throughput
    vec3* total_throughputs;
    checkCudaErrors(hipMalloc(&total_throughputs, sizeof(vec3) * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMemset(total_throughputs, 0.f, sizeof(vec3) * SCREEN_HEIGHT * SCREEN_WIDTH));

    // Sample through each pixel SAMPLES_PER_PIXEL times
    for (int i = 0; i < SAMPLES_PER_PIXEL; i++){

        // Initialise the rays
        initialise_ray<<<this->num_blocks, this->block_size>>>(
            d_rand_state,
            device_camera,
            ray_locations_device,
            ray_directions_device,
            ray_terminated_device,
            ray_throughputs_device,
            ray_bounces_device
        );
        checkCudaErrors(hipDeviceSynchronize());

        // Create bool to determine if all rays in the batch have collided with a light
        int rays_finished = 0; /* If not updated to false by trace_ray, end loop */
        int* device_rays_finished;
        checkCudaErrors(hipMalloc(&device_rays_finished, sizeof(int)));
        checkCudaErrors(hipMemset(device_rays_finished, 1, sizeof(int)));

        // Trace ray paths until all have intersected with a light/nothing
        unsigned int bounces = 0;
        while(rays_finished == 0 && bounces < MAX_RAY_BOUNCES){

            printf("Bounces: %d\n",bounces);

            // DIRECTION UPDATE
            // Don't modify the direction of the initial ray from the camera
            if (bounces > 0){

                // sample_next_ray_directions_randomly<<<this->num_blocks, this->block_size>>>(
                //     d_rand_state,
                //     ray_normals_device, 
                //     ray_directions_device,
                //     ray_throughputs_device,
                //     ray_terminated_device
                // );

                // Copy over the ray locations from the device to the host for inference
                float* ray_locations_host = new float[ SCREEN_HEIGHT * SCREEN_WIDTH * 3 ];
                checkCudaErrors(hipMemcpy(ray_locations_host, ray_locations_device, sizeof(float) * SCREEN_HEIGHT * SCREEN_WIDTH * 3, hipMemcpyDeviceToHost));

                // For each ray, compute the Q-values and importance sample a direction over them
                for (int b = 0; b < this->num_batches; b++){

                    // Compute the current batch size
                    int batch_start_idx = b*this->batch_size;
                    int current_batch_size = std::min(SCREEN_HEIGHT*SCREEN_WIDTH - batch_start_idx, this->batch_size);

                    // Initialise the Q-value storage on device
                    float* device_q_values;
                    checkCudaErrors(hipMalloc(&device_q_values, sizeof(float) * current_batch_size * GRID_RESOLUTION * GRID_RESOLUTION));

                    // Initialise the computational graph
                    dynet::ComputationGraph graph;

                    // Get the input expression 
                    std::vector<float> positions(current_batch_size*3);
                    memcpy(&(positions[0]), &(ray_locations_host[batch_start_idx*3]), sizeof(float)*current_batch_size*3);
                    dynet::Expression input = dynet::input(graph, dynet::Dim({3}, current_batch_size), positions);

                    // Get the q-vals
                    dynet::Expression prediction = this->dqn.network_inference(graph, input, false);
                    std::vector<float> q_vals = dynet::as_vector( graph.forward(prediction));                 // Some q_vals are all zero
                    
                    // Copy q-values to device
                    checkCudaErrors(hipMemcpy(device_q_values, &(q_vals[0]), sizeof(float) * q_vals.size(), hipMemcpyHostToDevice));
                
                    // Run cuda kernel to compute new ray directions
                    int threads = 16;
                    int blocks = int(current_batch_size/threads);
                    importance_sample_ray_directions<<<blocks, threads>>>(
                        d_rand_state,
                        device_q_values,
                        ray_normals_device,
                        ray_directions_device,
                        ray_locations_device,
                        ray_throughputs_device,
                        ray_terminated_device,
                        (b*this->batch_size)
                    );
                    hipDeviceSynchronize();

                    // Free memory
                    hipFree(device_q_values);
                }

                delete [] ray_locations_host;
            }

            // TRACE RAYS
            trace_ray<<<this->num_blocks, this->block_size>>>(
                device_scene,
                device_rays_finished,
                ray_locations_device, 
                ray_normals_device, 
                ray_directions_device,
                ray_terminated_device,  
                ray_throughputs_device,
                ray_bounces_device,
                bounces
            );
            hipDeviceSynchronize();

            // Copy over value to check if all rays have intersected with a light
            checkCudaErrors(hipMemcpy(&rays_finished, device_rays_finished, sizeof(int), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemset(device_rays_finished, 1, sizeof(int)));

            // Increment the number of bounces
            bounces++;            
        }

        // Add computed throughput values to the running total
        update_total_throughput<<<this->num_blocks, this->block_size>>>(
            ray_throughputs_device,
            total_throughputs
        );
        hipDeviceSynchronize();
        hipFree(device_rays_finished);
    }
    // Update the device_buffer with the throughput
    update_device_buffer<<<this->num_blocks, this->block_size>>>(
        device_buffer,
        total_throughputs
    );
    hipDeviceSynchronize();
    hipFree(total_throughputs);

    // Calculate the average path length
    int* total_path_lengths_device;
    checkCudaErrors(hipMalloc(&total_path_lengths_device, sizeof(int)));
    checkCudaErrors(hipMemset(total_path_lengths_device, 0, sizeof(int)));
    sum_path_lengths<<<this->num_blocks, this->block_size>>>(
        total_path_lengths_device,
        ray_bounces_device
    );
    int total_path_lengths = 0;
    checkCudaErrors(hipMemcpy(&total_path_lengths, total_path_lengths_device, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "Avg Path Length: " << total_path_lengths/(SCREEN_HEIGHT*SCREEN_WIDTH) << std::endl;    
}

// Gets the initial direction to shoot a ray in
__global__
void initialise_ray(
    hiprandState* d_rand_state,
    Camera* device_camera, 
    float* ray_locations, 
    float* ray_directions,
    bool* ray_terminated, 
    float* ray_throughputs,
    unsigned int* ray_bounces
){

    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;

    // Randomly sample a ray within the pixel
    Ray r = Ray::sample_ray_through_pixel(d_rand_state, *device_camera, x, y);
    ray_locations[(i*3)    ] = r.start.x;
    ray_locations[(i*3) + 1] = r.start.y;
    ray_locations[(i*3) + 2] = r.start.z;
    ray_directions[(i*3)    ] = r.direction.x;
    ray_directions[(i*3) + 1] = r.direction.y;
    ray_directions[(i*3) + 2] = r.direction.z;

    // Initialise ray_variables
    ray_terminated[i] = false;
    ray_throughputs[(i*3)    ] = 1.f;
    ray_throughputs[(i*3) + 1] = 1.f;
    ray_throughputs[(i*3) + 2] = 1.f;
    ray_bounces[i] = MAX_RAY_BOUNCES;
}

// Trace a ray for all ray locations given in the angles specified within the scene
__global__
void trace_ray(
    Scene* scene,
    int* rays_finished,
    float* ray_locations, 
    float* ray_normals, 
    float* ray_directions,
    bool* ray_terminated,  
    float* ray_throughputs,
    unsigned int* ray_bounces,
    int bounces
){
    
    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;

    // Do nothing if we have already intersected with the light
    if (ray_terminated[i] == true){
        return;
    }

    // For the current ray, get its next state by shooting a ray in the direction stored in ray_directions
    vec3 position = vec3(ray_locations[(i*3)], ray_locations[(i*3)+1], ray_locations[(i*3)+2]);
    vec3 dir = vec3(ray_directions[(i*3)], ray_directions[(i*3)+1], ray_directions[(i*3)+2]);

    // Create the ray and trace it
    Ray ray(vec4(position + (dir * 0.00001f), 1.f), vec4(dir, 1.f));
    ray.closest_intersection(scene);

    // Update position, normal, and discount factor based on intersection
    switch(ray.intersection.intersection_type){

        // TERMINAL STATE: R_(t+1) = Environment light power
        case NOTHING:
            ray_terminated[i] = true;
            ray_throughputs[(i*3)] = ray_throughputs[(i*3)] * ENVIRONMENT_LIGHT*1;
            ray_throughputs[(i*3)+1] = ray_throughputs[(i*3)+1] * ENVIRONMENT_LIGHT*1;
            ray_throughputs[(i*3)+2] = ray_throughputs[(i*3)+2] * ENVIRONMENT_LIGHT*1;
            ray_bounces[i] = (unsigned int)bounces;
            break;
        
        // TERMINAL STATE: R_(t+1) = Area light power
        case AREA_LIGHT:
            ray_terminated[i] = true;
            
            vec3 diffuse_p = scene->area_lights[ray.intersection.index].diffuse_p;
            ray_throughputs[(i*3)] = ray_throughputs[(i*3)] * diffuse_p.x;
            ray_throughputs[(i*3)+1] = ray_throughputs[(i*3)+1] * diffuse_p.y;
            ray_throughputs[(i*3)+2] = ray_throughputs[(i*3)+2] * diffuse_p.z;
            ray_bounces[i] = (unsigned int)bounces;
            break;

        // NON-TERMINAL STATE: R_(t+1) + \gamma * max_a Q(S_t+1, a) 
        // where  R_(t+1) = 0 for diffuse surfaces
        case SURFACE:
            vec3 new_loc = vec3(ray.intersection.position);
            ray_locations[(i*3)  ] = new_loc.x;
            ray_locations[(i*3)+1] = new_loc.y;
            ray_locations[(i*3)+2] = new_loc.z;

            vec3 new_norm = ray.intersection.normal;
            ray_normals[(i*3)  ] = new_norm.x; 
            ray_normals[(i*3)+1] = new_norm.y;
            ray_normals[(i*3)+2] = new_norm.z;

            vec3 BRDF = scene->surfaces[ray.intersection.index].material.diffuse_c;

            // discount_factors holds cos_theta currently, update rgb throughput first
            ray_throughputs[(i*3)] = ray_throughputs[(i*3)] * (BRDF.x / (float)M_PI);
            ray_throughputs[(i*3)+1] = ray_throughputs[(i*3)+1] * (BRDF.y / (float)M_PI);
            ray_throughputs[(i*3)+2] = ray_throughputs[(i*3)+2] * (BRDF.z / (float)M_PI);

            // Still a ray being to bounce, so not finished
            atomicExch(rays_finished, 0);
            break;
    }
}

// Importance samples rays directions from Q-values
__global__
void importance_sample_ray_directions(
    hiprandState* d_rand_state,
    float* device_q_values,
    float* ray_normals_device,
    float* ray_directions_device,
    float* ray_locations_device,
    float* ray_throughputs_device,
    bool* ray_terminated_device,
    int batch_start_idx
){

    // Batch index
    int i =  blockIdx.x * blockDim.x + threadIdx.x;

    if ( batch_start_idx+i >= SCREEN_HEIGHT*SCREEN_WIDTH ) return;

    int q_start_idx = i * GRID_RESOLUTION * GRID_RESOLUTION;

     // Do nothing if we have already intersected with the light
     if (ray_terminated_device[batch_start_idx + i] == true){
        return;
    }

    // // Copy array onto local memory to speed-up processing
    // // Importance sample over Q_values
    float rv = hiprand_uniform(&d_rand_state[batch_start_idx + i]);
    int direction_idx = 0;
    float q_sum = 0.f;
    for (int n = 0; n < GRID_RESOLUTION*GRID_RESOLUTION; n++){ 

        q_sum += device_q_values[q_start_idx + n];
        // printf("%.5f\n",q_sum);
        if ( q_sum > rv ){
            direction_idx = n;
            break;
        }
    }

    // // Get max q-val
    // int max_q_index = 0;
    // int max_q = device_q_values[0];
    // // float summation = 0.f;
    // for (int n = 1; n < GRID_RESOLUTION*GRID_RESOLUTION; n++){
    //     float temp_q = device_q_values[q_start_idx + n];
    //     // summation += temp_q;
    //     if (temp_q > max_q){
    //         max_q_index = n;
    //         max_q = temp_q;
    //     }
    // }
    // int direction_idx = max_q_index;

    // if (max_q_index == GRID_RESOLUTION*GRID_RESOLUTION-1){
    //     printf("%.3f\n",device_q_values[q_start_idx + max_q_index]);
    // }

    // Convert the direction index sampled into an actual 3D direction
    sample_ray_for_grid_index(
        d_rand_state,
        direction_idx,
        ray_directions_device,
        ray_normals_device,
        ray_locations_device,
        ray_throughputs_device,
        (batch_start_idx + i)
    );
}
