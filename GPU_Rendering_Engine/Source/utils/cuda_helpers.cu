#include "hip/hip_runtime.h"
#include "cuda_helpers.cuh"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ 
void init_rand_state(hiprandState* d_rand_state, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if((x >= width) || (y >= height)) return;
    int pixel_index = x*height + y;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &d_rand_state[pixel_index]);
 }