#include "hip/hip_runtime.h"
#include "ray.cuh"
#include "surface.cuh"
#include "area_light.cuh"
#include <limits>

__device__
Ray::Ray(vec4 start, vec4 direction) {
    this->start = start;
    vec3 dir3 = normalize(vec3(direction));
    this->direction = vec4(dir3, 1);
    Intersection i;
    i.intersection_type = NOTHING;
    this->intersection = i;
}

__device__
void Ray::closest_intersection(Surface* surfaces, AreaLight* light_planes, int light_plane_count, int surfaces_count) {
    
    this->intersection.distance = 999999.f;

    // Find intersection with surface
    for (int i = 0; i < surfaces_count; i++) {
        bool return_val = this->intersects(i, surfaces);
        if(return_val){
            this->intersection.intersection_type = SURFACE;
        }
    }
    
    // Find intersection with area lights
    for (int i = 0; i < light_plane_count; i++) { //TODO: Enum on type of closest intersection
        bool return_val = this->intersects(i, light_planes);
        if(return_val){
            this->intersection.intersection_type = AREA_LIGHT;
        }
    }
}

// Tests whether the triangle intersects a ray, closer to the current closest intersection
__device__
bool Ray::intersects(int index, Surface* surfaces) {
    bool returnVal = false;
    vec4 start = this->start;
    vec4 dir = this->direction;

    vec4 v0 = surfaces[index].v0;
    vec4 v1 = surfaces[index].v1;
    vec4 v2 = surfaces[index].v2;

    vec3 e1 = vec3(v1.x - v0.x, v1.y - v0.y, v1.z - v0.z);
    vec3 e2 = vec3(v2.x - v0.x, v2.y - v0.y, v2.z - v0.z);
    vec3 b = vec3(start.x - v0.x, start.y - v0.y, start.z - v0.z);

    dir = vec4(vec3(dir) * (float)SCREEN_HEIGHT, 1);

    mat3 A(vec3(-dir), e1, e2);

    // solution.x = t: Scalar position of intersection along raw
    // solution.y = u: Scalar position along vecotr (v1- v0)
    // solution.z = v: Scalar position along vecotr (v2- v0)
    vec3 solution;
    bool crmr = cramer(A, b, solution);

    if (crmr && solution.x >= 0.0f && solution.y >= 0.0f && solution.z >= 0.0f && solution.y + solution.z <= 1.0f) {
        if (solution.x < this->intersection.distance + EPS && solution.x > EPS) {
            this->intersection.position = start + solution.x * dir;
            this->intersection.position[3] = 1;
            this->intersection.distance = solution.x;
            this->intersection.normal = surfaces[index].normal;
            this->intersection.index = index;
            returnVal = true;
        }
    }
    return returnVal;
}

// Tests whether the triangle intersects a ray, closer to the current closest intersection
__device__
bool Ray::intersects(int index, AreaLight* area_lights) {
    bool returnVal = false;
    vec4 start = this->start;
    vec4 dir = this->direction;

    vec4 v0 = area_lights[index].v0;
    vec4 v1 = area_lights[index].v1;
    vec4 v2 = area_lights[index].v2;

    vec3 e1 = vec3(v1.x - v0.x, v1.y - v0.y, v1.z - v0.z);
    vec3 e2 = vec3(v2.x - v0.x, v2.y - v0.y, v2.z - v0.z);
    vec3 b = vec3(start.x - v0.x, start.y - v0.y, start.z - v0.z);

    dir = vec4(vec3(dir) * (float)SCREEN_HEIGHT, 1);

    mat3 A(vec3(-dir), e1, e2);

    // solution.x = t: Scalar position of intersection along raw
    // solution.y = u: Scalar position along vecotr (v1- v0)
    // solution.z = v: Scalar position along vecotr (v2- v0)
    vec3 solution;
    bool crmr = cramer(A, b, solution);

    if (crmr && solution.x >= 0.0f && solution.y >= 0.0f && solution.z >= 0.0f && solution.y + solution.z <= 1.0f) {
        if (solution.x < this->intersection.distance + EPS && solution.x > EPS) {
            this->intersection.position = start + solution.x * dir;
            this->intersection.position[3] = 1;
            this->intersection.distance = solution.x;
            this->intersection.normal = area_lights[index].normal;
            this->intersection.index = index;
            returnVal = true;
        }
    }
    return returnVal;
}

// Cramers Rule: Solve a 3x3 linear equation system
__device__ 
bool Ray::cramer(mat3 A, vec3 b, vec3& solution) {
    bool ret = false;
    // Initialise the solution output
    solution = vec3(0,0,0);
    float detA = determinant(A);
    if (detA != 0) {
        ret = true;
        // Temp variable to hold the value of A
        mat3 temp = A;

        A[0] = b;
        solution.x = determinant(A) / detA;
        A = temp;

        A[1] = b;
        solution.y = determinant(A) / detA;
        A = temp;

        A[2] = b;
        solution.z = determinant(A) / detA;
        A = temp;
    } else {
        ret = false;
    }
    return ret;
}


// Rotate a ray by "yaw"
__device__
void Ray::rotate_ray(float yaw) {
    mat4 R = mat4(1.0);
    R[0] = vec4(cos(yaw), 0, sin(yaw), 0);
    R[2] = vec4(-sin(yaw), 0, cos(yaw), 0);
    this->direction = (R * this->direction);
}

