#include "hip/hip_runtime.h"
#include "neural_q_pathtracer.cuh"


__host__
NeuralQPathtracer::NeuralQPathtracer(
        unsigned int frames,
        unsigned int batch_size,
        SDLScreen& screen,
        Scene& scene,
        Camera& camera,
        int argc,
        char** argv
    ){

    //////////////////////////////////////////////////////////////
    /*                  Assign attributes                       */
    //////////////////////////////////////////////////////////////
    this->ray_batch_size = batch_size; /* How many rays to be processed at once */
    this->num_batches = int((SCREEN_HEIGHT * SCREEN_WIDTH)/batch_size) + 1; /* How many batches in total */
    printf("Batch Size: %d\n", batch_size);
    printf("Number of Batches: %d\n", num_batches);

    dim3 b_size(8,8);
    this->block_size = b_size; /* How many threads in a single block to process the screen*/
    int blocks_x = (SCREEN_WIDTH + this->block_size.x - 1)/this->block_size.x;
    int blocks_y = (SCREEN_HEIGHT + this->block_size.y - 1)/this->block_size.y;
    dim3 n_bs(blocks_x, blocks_y);
    this->num_blocks = n_bs;/* How many blocks to process all pixels on the screen */

    //////////////////////////////////////////////////////////////
    /*                Initialise the DQN                        */
    //////////////////////////////////////////////////////////////
    //TODO: Might have to specify the amount of memory the GPU can use
    // beforehand, otherwise it seems to assign over memory allocated later
    // on. It may continue to do this when calculating back&forwad prop
    auto dyparams = dynet::extract_dynet_params(argc, argv);
    dynet::initialize(dyparams);
    dynet::ParameterCollection model;
    dynet::AdamTrainer trainer(model);
    this->dqn = DQNetwork();
    this->dqn.initialize(model);

    //////////////////////////////////////////////////////////////
    /*          Intialise Pixel value buffers                   */
    //////////////////////////////////////////////////////////////
    vec3* host_buffer = new vec3[ SCREEN_HEIGHT * SCREEN_WIDTH ];
    vec3* device_buffer;
    checkCudaErrors(hipMalloc(&device_buffer, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(vec3)));

    //////////////////////////////////////////////////////////////
    /*          Initialise Prev Host buffers                    */
    //////////////////////////////////////////////////////////////
    float* prev_location_host = new float[ SCREEN_HEIGHT * SCREEN_WIDTH * 3 ];
    unsigned int* directions_host = new unsigned int[ SCREEN_HEIGHT * SCREEN_WIDTH ];
    
    //////////////////////////////////////////////////////////////
    /*          Initialise ray arrays on CUDA device            */
    //////////////////////////////////////////////////////////////
    float* ray_locations;   /* Ray intersection location (State) */
    float* ray_normals;     /* Intersection normal */
    float* ray_directions;  /* Direction to next shoot the ray */
    bool* ray_terminated;  /* Has the ray intersected with a light/nothing */
    float* ray_rewards;    /* Reward recieved from Q(s,a) */
    float* ray_discounts;  /* Discount factor for current rays path */
    float* ray_throughputs; /* Throughput for calc pixel value */
    unsigned int* ray_bounces; /* Total number of bounces for each ray before intersection*/

    checkCudaErrors(hipMalloc(&ray_locations, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_normals, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_directions, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_terminated, sizeof(bool) * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_rewards, sizeof(float) * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_discounts, sizeof(float) * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_throughputs, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMalloc(&ray_bounces, sizeof(unsigned int) *SCREEN_HEIGHT *SCREEN_WIDTH));
    
    Camera* device_camera; /* Camera on the CUDA device */
    Surface* device_surfaces;
    AreaLight* device_light_planes;
    Scene* device_scene;   /* Scenes to render */

    // Copy the camera
    checkCudaErrors(hipMalloc(&device_camera, sizeof(Camera)));
    checkCudaErrors(hipMemcpy(device_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice));
    
    // Copy surfaces into device memory space
    checkCudaErrors(hipMalloc(&device_surfaces, scene.surfaces_count * sizeof(Surface)));
    checkCudaErrors(hipMemcpy(device_surfaces, scene.surfaces, scene.surfaces_count * sizeof(Surface), hipMemcpyHostToDevice));

    // Copy light planes into device memory space
    checkCudaErrors(hipMalloc(&device_light_planes, scene.area_light_count * sizeof(AreaLight)));
    checkCudaErrors(hipMemcpy(device_light_planes, scene.area_lights, scene.area_light_count * sizeof(AreaLight), hipMemcpyHostToDevice));    

    // Copy the scene structure into the device and its corresponding pointers to Surfaces and Area Lights
    checkCudaErrors(hipMalloc(&device_scene, sizeof(Scene)));
    checkCudaErrors(hipMemcpy(device_scene, &scene, sizeof(Scene), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&(device_scene->surfaces), &device_surfaces, sizeof(Surface*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&(device_scene->area_lights), &device_light_planes, sizeof(AreaLight*), hipMemcpyHostToDevice));    
    
    //////////////////////////////////////////////////////////////
    /*                  Intialise cuRand State                  */
    //////////////////////////////////////////////////////////////
    hiprandState * d_rand_state;
    checkCudaErrors(hipMalloc(&d_rand_state, (float)SCREEN_HEIGHT * (float)SCREEN_WIDTH * sizeof(hiprandState)));
    init_rand_state<<<this->num_blocks, this->block_size>>>(d_rand_state, SCREEN_WIDTH, SCREEN_HEIGHT);

    //////////////////////////////////////////////////////////////
    /*                  Render the frames                       */
    //////////////////////////////////////////////////////////////
    for (int i = 0; i < frames; i++){
        //Clear the pixel buffer
        memset(host_buffer, 0.f, sizeof(vec3)* SCREEN_HEIGHT * SCREEN_WIDTH);

        /* Compute frame time */
        static int t = SDL_GetTicks();
        int t2 = SDL_GetTicks();
        float dt = float(t2-t);
        t = t2;
        printf("Render Time: %.3f ms.\n", dt);
        
        // Fill the pixel buffer each frame using Deep Q-Learning strategy
        this->render_frame(
            trainer, 
            d_rand_state,
            device_camera,
            device_scene,
            device_buffer,
            prev_location_host,
            directions_host,
            ray_locations,
            ray_normals,   
            ray_directions,
            ray_terminated,  
            ray_rewards,   
            ray_discounts, 
            ray_throughputs,
            ray_bounces
        );

        // Copy the device buffer values to the host buffer
        checkCudaErrors(hipMemcpy(host_buffer, device_buffer, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(vec3), hipMemcpyDeviceToHost));

        // Display the rendered frame
        for (int x = 0; x < SCREEN_WIDTH; x++){
            for (int y = 0; y < SCREEN_HEIGHT; y++){
                screen.PutPixelSDL(x, y, host_buffer[x*(int)SCREEN_HEIGHT + y]);
            }
        }
        screen.SDL_Renderframe();
    }

    //////////////////////////////////////////////////////////////
    /*          Save the image and kill the screen              */
    //////////////////////////////////////////////////////////////
    screen.SDL_SaveImage("../Images/render.bmp");
    screen.kill_screen();

    //////////////////////////////////////////////////////////////
    /*                      Free memory used                    */
    //////////////////////////////////////////////////////////////
    delete [] host_buffer;
    delete [] prev_location_host;
    delete [] directions_host;
    hipFree(device_buffer);
    hipFree(d_rand_state);
    hipFree(ray_locations);
    hipFree(ray_normals);
    hipFree(ray_directions);
    hipFree(ray_terminated);
    hipFree(ray_rewards);
    hipFree(ray_throughputs);
    hipFree(ray_bounces);
    hipFree(device_camera);
    hipFree(device_surfaces);
    hipFree(device_light_planes);
    hipFree(device_scene);
}

__host__
void NeuralQPathtracer::render_frame(
        dynet::AdamTrainer trainer,
        hiprandState* d_rand_state,
        Camera* device_camera,
        Scene* device_scene,
        vec3* device_buffer,
        float* prev_location_host,
        unsigned int* directions_host,
        float* ray_locations,   /* Ray intersection location (State) */
        float* ray_normals,     /* Intersection normal */
        float* ray_directions,  /* Direction to next shoot the ray */
        bool* ray_terminated,  /* Has the ray intersected with a light/nothing */
        float* ray_rewards,    /* Reward recieved from Q(s,a) */
        float* ray_discounts,  /* Discount factor for current rays path */
        float* ray_throughputs,  /* Throughput for calc pixel value */
        unsigned int* ray_bounces /* Total number of bounces for each ray before intersection*/
    ){

    // Initialise the computation graph
    dynet::ComputationGraph graph;

    // Initialise buffer to hold total throughput
    vec3* total_throughputs;
    checkCudaErrors(hipMalloc(&total_throughputs, sizeof(vec3) * SCREEN_HEIGHT * SCREEN_WIDTH));
    checkCudaErrors(hipMemset(total_throughputs, 0.f, sizeof(vec3) * SCREEN_HEIGHT * SCREEN_WIDTH));
    
    // Sample through each pixel SAMPLES_PER_PIXEL times
    for (int i = 0; i < SAMPLES_PER_PIXEL; i++){
        // Initialise mini-batch ray variables
        initialise_ray<<<this->num_blocks, this->block_size>>>(
            d_rand_state,
            device_camera, 
            ray_locations, 
            ray_directions,
            ray_terminated, 
            ray_rewards, 
            ray_discounts,
            ray_throughputs,
            ray_bounces
        );
        checkCudaErrors(hipDeviceSynchronize());

        // Create bool to determine if all rays in the batch have collided with a light
        int rays_finished = 0; /* If not updated to false by trace_ray, end loop */
        int* device_rays_finished;
        checkCudaErrors(hipMalloc(&device_rays_finished, sizeof(int)));
        checkCudaErrors(hipMemset(device_rays_finished, 1, sizeof(int)));

        // Trace batch rays path until all have intersected with a light
        unsigned int bounces = 0;
        float loss = 0.f;
        while(rays_finished == 0 && bounces < MAX_RAY_BOUNCES){

            printf("Bounce: %d/%d\n", bounces, MAX_RAY_BOUNCES);

            // Maintain previous locations for reinforcment Q(s,a) update
            checkCudaErrors(hipMemcpy(prev_location_host, ray_locations, sizeof(vec3) * SCREEN_HEIGHT * SCREEN_WIDTH, hipMemcpyDeviceToHost));

            // Does not apply to shooting from camera
            if (bounces > 0){

                // Device index values
                unsigned int* directions_device;
                checkCudaErrors(hipMalloc(&directions_device, sizeof(unsigned int) * SCREEN_HEIGHT*SCREEN_WIDTH));

                // For each batch sample Q-values and apply eta-greedy policy
                for(int n = 0; n < this->num_batches; n++){
                    
                    // Compute Batch-Size
                    unsigned int current_batch_size = std::min(SCREEN_HEIGHT*SCREEN_WIDTH - (n*this->ray_batch_size), this->ray_batch_size);
                    if (current_batch_size < 1) break;

                    // Get Q-values
                    graph.clear();
                    dynet::Dim input_dim({3},current_batch_size);
                    std::vector<float> input_states(3*current_batch_size);
                    memcpy(&(input_states[0]), &prev_location_host[n*this->ray_batch_size*3], sizeof(float) * 3 * current_batch_size);
                    dynet::Expression states_batch = dynet::input(graph, input_dim, &input_states);
                    dynet::Expression current_qs_expr = this->dqn.network_inference(graph, states_batch, true);
                    std::vector<float> current_qs = dynet::as_vector(current_qs_expr.value());

                    // Copy Q-vals to GPU for find the argmax
                    float* current_qs_device;
                    checkCudaErrors(hipMalloc(&current_qs_device, sizeof(float) * current_qs.size()));
                    checkCudaErrors(hipMemcpy(current_qs_device, &(current_qs[0]), sizeof(float) * current_qs.size() , hipMemcpyHostToDevice));

                    // Get direction indices (Call once for every element in the batch)
                    int threads = 32;
                    int blocks = int(current_batch_size/32)+1;
                    sample_batch_ray_indices_eta_greedy<<<threads, blocks>>>(
                        ETA,
                        d_rand_state,
                        directions_device,
                        current_qs_device,
                        n,
                        this->ray_batch_size
                    );
                    hipDeviceSynchronize();

                    // Free memory
                    hipFree(current_qs_device);
                }

                // Copy sampled indices (actions) back to host to perform backprop with
                checkCudaErrors(hipMemcpy(directions_host, directions_device, sizeof(unsigned int) * SCREEN_HEIGHT * SCREEN_WIDTH, hipMemcpyDeviceToHost));
                
                // Sample the ray direction from the calculated indices
                sample_ray_for_grid_index<<<this->num_blocks, this->block_size>>>(
                    d_rand_state,
                    directions_device,
                    ray_directions,
                    ray_locations,
                    ray_normals,
                    ray_throughputs,
                    ray_terminated
                );
                hipDeviceSynchronize();

                // Free memory
                hipFree(directions_device);
            }

            // Trace the rays in their set directions
            trace_ray<<<this->num_blocks, this->block_size>>>(
                device_scene,
                device_rays_finished,
                ray_locations, 
                ray_normals,
                ray_directions, 
                ray_terminated, 
                ray_rewards,
                ray_discounts,
                ray_throughputs,
                ray_bounces,
                bounces
            );  
            hipDeviceSynchronize();

            // Does not apply to shooting from camera //TODO: We are current backpropogating rays that have terminated continually, this is bad
            if(bounces > 0){

                // Copy data from Cuda device to host for usage
                float* ray_locations_host = new float[ SCREEN_HEIGHT * SCREEN_WIDTH * 3 ];
                checkCudaErrors(hipMemcpy(ray_locations_host, ray_locations, sizeof(float) * 3 * SCREEN_HEIGHT * SCREEN_WIDTH , hipMemcpyDeviceToHost));
                // Run learning rule on the network with the results received and sample new direction for each ray in batches
                for(int n = 0; n < this->num_batches; n++){
                    
                    graph.clear();

                    // 1) Create the input expression to the neural network for S_t+1
                    unsigned int current_batch_size = std::min(SCREEN_HEIGHT*SCREEN_WIDTH - (n*this->ray_batch_size), this->ray_batch_size);
                    if (current_batch_size < 1) break;

                    dynet::Dim input_dim({3},current_batch_size);
                    std::vector<float> input_vals(3*current_batch_size);
                    memcpy(&(input_vals[0]), &ray_locations_host[n*current_batch_size*3], sizeof(float) * 3 * current_batch_size);
                    dynet::Expression input_batch = dynet::input(graph, input_dim, &input_vals);

                    // 2) Get max_a Q(S_{t+1}, a)
                    dynet::Expression next_qs = dynet::max_dim(this->dqn.network_inference(graph, input_batch, false),0);
                    std::vector<float> td_targets = dynet::as_vector(graph.forward(next_qs));

                    // 3) Compute TD-Targets
                    float* td_targets_device;
                    checkCudaErrors(hipMalloc(&td_targets_device, sizeof(float) * current_batch_size));
                    checkCudaErrors(hipMemcpy(td_targets_device, &(td_targets[0]), sizeof(float) * current_batch_size, hipMemcpyHostToDevice));

                    compute_td_targets<<<int(current_batch_size/32)+1, 32>>>(
                        td_targets_device,
                        ray_rewards,
                        ray_discounts
                    );
                    hipDeviceSynchronize();
                    checkCudaErrors(hipMemcpy(&(td_targets[0]), td_targets_device, sizeof(float) * current_batch_size, hipMemcpyDeviceToHost));
                    hipFree(td_targets_device);

                    // 4) Reset computational graph and use target_value as a constant
                    graph.clear();
                    dynet::Expression td_target = dynet::input(graph, dynet::Dim({1}, current_batch_size), td_targets);

                    // // 5) Get current Q(s,a) value
                    std::vector<float> input_states(3*current_batch_size);
                    memcpy(&(input_states[0]), &prev_location_host[n*current_batch_size*3], sizeof(float) * 3 * current_batch_size);

                    dynet::Expression states_batch = dynet::input(graph, input_dim, &input_states);
                    dynet::Expression current_all_qs = this->dqn.network_inference(graph, states_batch, true);
                    
                    // Get the vector of action value indices we took 
                    std::vector<unsigned int> action_value_indices(current_batch_size);
                    memcpy(&action_value_indices[0], &directions_host[this->ray_batch_size*n], sizeof(unsigned int) * current_batch_size);

                    // Get the current Q values for the actions taken
                    dynet::Expression current_qs = dynet::pick(current_all_qs, action_value_indices, (unsigned int) 0);
                    
                    // // 6) Calculate the loss
                    dynet::Expression loss_expr = dynet::pow((td_target - current_qs), dynet::input(graph, 2.f));  
                    loss_expr = dynet::sum_batches(loss_expr); 
                    loss += dynet::as_scalar(graph.forward(loss_expr));

                    // // 7) Train the network
                    graph.backward(loss_expr);
                    trainer.update();
                }

                // Dete the host arrays
                delete [] ray_locations_host;
            }

            // Copy over value to check if all rays have intersected with a light
            checkCudaErrors(hipMemcpy(&rays_finished, device_rays_finished, sizeof(int), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemset(device_rays_finished, 1, sizeof(int)));

            // Increment the number of bounces
            bounces++;
        }
        printf("loss: %.3f\n",loss);

        // Add computed throughput values to the running total
        update_total_throughput<<<this->num_blocks, this->block_size>>>(
            ray_throughputs,
            total_throughputs
        );
        hipDeviceSynchronize();
        hipFree(device_rays_finished);
    }
    // Update the device_buffer with the throughput
    update_device_buffer<<<this->num_blocks, this->block_size>>>(
        device_buffer,
        total_throughputs
    );
    hipDeviceSynchronize();
    hipFree(total_throughputs);

    // Calculate the average path length
    int* total_path_lengths_device;
    checkCudaErrors(hipMalloc(&total_path_lengths_device, sizeof(int)));
    checkCudaErrors(hipMemset(total_path_lengths_device, 0, sizeof(int)));
    sum_path_lengths<<<this->num_blocks, this->block_size>>>(
        total_path_lengths_device,
        ray_bounces
    );
    int total_path_lengths = 0;
    checkCudaErrors(hipMemcpy(&total_path_lengths, total_path_lengths_device, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "Avg Path Length: " << total_path_lengths/(SCREEN_HEIGHT*SCREEN_WIDTH) << std::endl;
}

// Gets the initial direction to shoot a ray in
__global__
void initialise_ray(
        hiprandState* d_rand_state,
        Camera* device_camera, 
        float* ray_locations, 
        float* ray_directions,
        bool* ray_terminated, 
        float* ray_rewards, 
        float* ray_discounts,
        float* ray_throughputs,
        unsigned int* ray_bounces
    ){

    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;

    // Randomly sample a ray within the pixel
    Ray r = Ray::sample_ray_through_pixel(d_rand_state, *device_camera, x, y);
    ray_locations[(i*3)    ] = r.start.x;
    ray_locations[(i*3) + 1] = r.start.y;
    ray_locations[(i*3) + 2] = r.start.z;
    ray_directions[(i*3)    ] = r.direction.x;
    ray_directions[(i*3) + 1] = r.direction.y;
    ray_directions[(i*3) + 2] = r.direction.z;

    // Initialise ray_variables
    ray_rewards[i] = 0.f;
    ray_terminated[i] = false;
    ray_throughputs[(i*3)    ] = 1.f;
    ray_throughputs[(i*3) + 1] = 1.f;
    ray_throughputs[(i*3) + 2] = 1.f;
    ray_discounts[i] = 1.f;
    ray_bounces[i] = MAX_RAY_BOUNCES;
}

// Trace a ray for all ray locations given in the angles specified within the scene
__global__
void trace_ray(
        Scene* scene,
        int* rays_finished,
        float* ray_locations, 
        float* ray_normals, 
        float* ray_directions,
        bool* ray_terminated, 
        float* ray_rewards,
        float* ray_discounts, 
        float* ray_throughputs,
        unsigned int* ray_bounces,
        int bounces
    ){
    
    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;

    // Do nothing if we have already intersected with the light
    if (ray_terminated[i] == true){
        return;
    }

    // For the current ray, get its next state by shooting a ray in the direction stored in ray_directions
    vec3 position = vec3(ray_locations[(i*3)], ray_locations[(i*3)+1], ray_locations[(i*3)+2]);
    vec3 dir = vec3(ray_directions[(i*3)], ray_directions[(i*3)+1], ray_directions[(i*3)+2]);

    // Create the ray and trace it
    Ray ray(vec4(position + (dir * 0.00001f), 1.f), vec4(dir, 1.f));
    ray.closest_intersection(scene);

    // Update position, normal, and discount factor based on intersection
    switch(ray.intersection.intersection_type){

        // TERMINAL STATE: R_(t+1) = Environment light power
        case NOTHING:
            ray_terminated[i] = true;
            ray_rewards[i] = ENVIRONMENT_LIGHT;
            ray_throughputs[(i*3)] = ray_throughputs[(i*3)] * ENVIRONMENT_LIGHT;
            ray_throughputs[(i*3)+1] = ray_throughputs[(i*3)+1] * ENVIRONMENT_LIGHT;
            ray_throughputs[(i*3)+2] = ray_throughputs[(i*3)+2] * ENVIRONMENT_LIGHT;
            ray_bounces[i] = (unsigned int)bounces;
            break;
        
        // TERMINAL STATE: R_(t+1) = Area light power
        case AREA_LIGHT:
            ray_terminated[i] = true;
            float diffuse_light_power = scene->area_lights[ray.intersection.index].luminance; 
            ray_rewards[i] = diffuse_light_power;
            
            vec3 diffuse_p = scene->area_lights[ray.intersection.index].diffuse_p;
            ray_throughputs[(i*3)] = ray_throughputs[(i*3)] * diffuse_p.x;
            ray_throughputs[(i*3)+1] = ray_throughputs[(i*3)+1] * diffuse_p.y;
            ray_throughputs[(i*3)+2] = ray_throughputs[(i*3)+2] * diffuse_p.z;
            ray_bounces[i] = (unsigned int)bounces;
            break;

        // NON-TERMINAL STATE: R_(t+1) + \gamma * max_a Q(S_t+1, a) 
        // where  R_(t+1) = 0 for diffuse surfaces
        case SURFACE:
            vec3 new_loc = vec3(ray.intersection.position);
            ray_locations[(i*3)  ] = new_loc.x;
            ray_locations[(i*3)+1] = new_loc.y;
            ray_locations[(i*3)+2] = new_loc.z;

            vec3 new_norm = ray.intersection.normal;
            ray_normals[(i*3)  ] = new_norm.x; 
            ray_normals[(i*3)+1] = new_norm.y;
            ray_normals[(i*3)+2] = new_norm.z;

            vec3 BRDF = scene->surfaces[ray.intersection.index].material.diffuse_c;
            
            // Get luminance of material
            float max_rgb = max(BRDF.x, BRDF.y);
            max_rgb = max(BRDF.z, max_rgb);
            float min_rgb = min(BRDF.x, BRDF.y);
            min_rgb = min(BRDF.z, min_rgb);
            float luminance = 0.5f * (max_rgb + min_rgb);

            // discount_factors holds cos_theta currently, update rgb throughput first
            ray_throughputs[(i*3)] = ray_throughputs[(i*3)] * (BRDF.x / (float)M_PI);
            ray_throughputs[(i*3)+1] = ray_throughputs[(i*3)+1] * (BRDF.y / (float)M_PI);
            ray_throughputs[(i*3)+2] = ray_throughputs[(i*3)+2] * (BRDF.z / (float)M_PI);

            // Now update discount_factors with luminance
            ray_discounts[i] *= luminance;
            // Still a ray being to bounce, so not finished
            atomicExch(rays_finished, 0);
            break;
    }
}

// Sample random directions to further trace the rays in
__global__
void sample_next_ray_directions_randomly(
        hiprandState* d_rand_state,
        float* ray_normals, 
        float* ray_directions,
        float* ray_throughputs,
        bool* ray_terminated
    ){
    
    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;

    // Do nothing if we have already intersected with the light
    if (ray_terminated[i] == true){
        return;
    }

    // Sample the new direction and record it along with cos_theta
    float cos_theta;
    vec3 dir = vec3(sample_random_direction_around_intersection(d_rand_state, vec3(ray_normals[(i*3)], ray_normals[(i*3)+1], ray_normals[(i*3)+2]), cos_theta));
    ray_directions[(i*3)    ] = dir.x;
    ray_directions[(i*3) + 1] = dir.y;
    ray_directions[(i*3) + 2] = dir.z;

    // Update throughput with new sampled angle
    ray_throughputs[(i*3)    ] = (ray_throughputs[(i*3)    ] * cos_theta)/RHO;
    ray_throughputs[(i*3) + 1] = (ray_throughputs[(i*3) + 1] * cos_theta)/RHO;
    ray_throughputs[(i*3) + 2] = (ray_throughputs[(i*3) + 2] * cos_theta)/RHO;
}

// Sample index directions according the neural network q vals
__global__
void sample_batch_ray_indices_eta_greedy(
        float eta,
        hiprandState* d_rand_state,
        unsigned int* directions_device,
        float* current_qs_device,
        int batch_index,
        int batch_size
    ){
        // Get the index of the ray in the current batch
        int i =  blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= batch_size) return;
        
        // Sample the random number to be used for eta-greedy policy
        float rv = hiprand_uniform(&d_rand_state[batch_size*batch_index + i]);

        // The total number of actions to choose from
        int action_count = GRID_RESOLUTION*GRID_RESOLUTION;

        // Greedy
        if (rv > eta){
            // Get the larget q-values index
            unsigned int max_idx = 0;
            float max_q = current_qs_device[(action_count)*i];
            for (unsigned int n = 0; n < action_count; n++){
                if (current_qs_device[(action_count)*i + n] > max_q){
                    max_idx = n;
                    max_q = current_qs_device[(action_count)*i + n];
                }
            }
            // Update the indices of directions with max_idx
            directions_device[batch_size*batch_index + i] = max_idx;
        }
        // Explore
        else{
            // Sample a random grid index
            directions_device[batch_size*batch_index + i] = 
                (unsigned int)(int((hiprand_uniform(&d_rand_state[batch_size*batch_index + i]) - 0.0001f) * action_count));
        }
}

// Randomly sample with the given grid index a 3D ray direction
__global__
void sample_ray_for_grid_index(
    hiprandState* d_rand_state,
    unsigned int* grid_indices,
    float* ray_directions,
    float* ray_locations,
    float* ray_normals,
    float* ray_throughputs,
    bool* ray_terminated
){
    // // Ray Index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;

    // Do nothing if we have already intersected with the light
    if (ray_terminated[i] == true){
        return;
    }

    // Convert the index to a grid position
    int dir_grid = (int)grid_indices[i];
    int dir_x = int(dir_grid/GRID_RESOLUTION);
    int dir_y = dir_grid - (dir_x*GRID_RESOLUTION);

    // Convert to 3D direction and update the direction
    vec3 position = vec3(ray_locations[(i*3)], ray_locations[(i*3) + 1], ray_locations[(i*3) + 2]);
    vec3 normal  = vec3(ray_normals[(i*3)], ray_normals[(i*3) + 1], ray_normals[(i*3) + 2]);
    mat4 transformation_matrix = create_transformation_matrix(normal, vec4(position, 1.f));
    vec3 dir = convert_grid_pos_to_direction_random(d_rand_state, (float) dir_x, (float) dir_y, i, position, transformation_matrix);
    ray_directions[(i*3)    ] = dir.x;
    ray_directions[(i*3) + 1] = dir.y;
    ray_directions[(i*3) + 2] = dir.z;

    // Update throughput with new sampled angle
    float cos_theta = dot(normal, dir);
    ray_throughputs[(i*3)    ] = (ray_throughputs[(i*3)    ] * cos_theta)/RHO;
    ray_throughputs[(i*3) + 1] = (ray_throughputs[(i*3) + 1] * cos_theta)/RHO;
    ray_throughputs[(i*3) + 2] = (ray_throughputs[(i*3) + 2] * cos_theta)/RHO;
}

// Compute the TD targets for the current batch size
__global__
void compute_td_targets(
        float* td_targets_device,
        float* ray_rewards,
        float* ray_discounts
    ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    td_targets_device[i] =  ray_rewards[i] + td_targets_device[i]*ray_discounts[i];
}   

// Update pixel values stored in the device_buffer
__global__
void update_total_throughput(
        float* ray_throughputs,
        vec3* total_throughputs
    ){

    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;
    
    // Accumulate
    total_throughputs[i] += vec3(ray_throughputs[(i*3)], ray_throughputs[(i*3)+1], ray_throughputs[(i*3)+2]);
}

// Update the device_buffer with the throughput
__global__
void update_device_buffer(
    vec3* device_buffer,
    vec3* total_throughputs
    ){
    
    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;

    // Update
    device_buffer[i] = total_throughputs[i]/(float)SAMPLES_PER_PIXEL;
}

// Sum up all path lengths
__global__
void sum_path_lengths(
    int* total_path_lengths_device,
    unsigned int* ray_bounces
){
    // Ray index
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    int i = SCREEN_HEIGHT*x + y;

    atomicAdd(total_path_lengths_device, (int)ray_bounces[i]); 
}