#include "hip/hip_runtime.h"
#include "voronoi_trace.cuh"
#include <iostream>

__global__
void draw_voronoi_trace(vec3* device_buffer, hiprandState* d_rand_state, RadianceMap* radiance_map, Camera* camera, Scene* scene){

    // Populate the shared GPU/CPU screen buffer
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    device_buffer[ x*SCREEN_HEIGHT + y ] = voronoi_trace(d_rand_state, camera, radiance_map, x, y, scene);
}

__device__
vec3 voronoi_trace(hiprandState* d_rand_state, Camera* camera, RadianceMap* radiance_map, int pixel_x, int pixel_y, Scene* scene){
        // Generate the random point within a pixel for the ray to pass through
        float x = (float)pixel_x + hiprand_uniform(&d_rand_state[pixel_x*(int)SCREEN_HEIGHT + pixel_y]);
        float y = (float)pixel_y + hiprand_uniform(&d_rand_state[pixel_x*(int)SCREEN_HEIGHT + pixel_y]);
        // Set direction to pass through pixel (pixel space -> Camera space)
        vec4 dir((x - (float)SCREEN_WIDTH / 2.f) , (y - (float)SCREEN_HEIGHT / 2.f) , (float)FOCAL_LENGTH , 1);

        // Create a ray that we will change the direction for below
        Ray ray(camera->position, dir);
        ray.rotate_ray(camera->yaw);

        // Trace the path of the ray to find the closest intersection
        ray.closest_intersection(scene);

        if (ray.intersection.intersection_type == SURFACE){
            // Get the voronoi colour of the intersection point
            return radiance_map->get_voronoi_colour(ray.intersection);
        }
        else{
            return vec3(1.f);
        }
}