#include "hip/hip_runtime.h"
#include "reinforcement_path_tracing.cuh"
//cuRand
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__
void draw_reinforcement_path_tracing(vec3* device_buffer, hiprandState* d_rand_state, Camera camera, AreaLight* light_planes, Surface* surfaces, int light_plane_count, int surfaces_count){
    
    // Populate the shared GPU/CPU screen buffer
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Path trace the ray to find the colour to paint the pixel
    device_buffer[x*(int)SCREEN_HEIGHT + y] = path_trace_reinforcement(d_rand_state, camera, x, y, surfaces, light_planes, light_plane_count, surfaces_count);

}

__device__
vec3 path_trace_reinforcement(hiprandState* d_rand_state, Camera camera, int pixel_x, int pixel_y, Surface* surfaces, AreaLight* light_planes, int light_plane_count, int surfaces_count){
    vec3 irradiance = vec3(0.f);
    for (int i = 0; i < SAMPLES_PER_PIXEL; i++){
        
        Ray ray = Ray::sample_ray_through_pixel(d_rand_state, camera, pixel_x, pixel_y);

        // Trace the path of the ray
        irradiance += path_trace_reinforcement_iterative(d_rand_state, ray, surfaces, light_planes, light_plane_count, surfaces_count);
    }
    irradiance /= (float)SAMPLES_PER_PIXEL;
    return irradiance;
}

__device__
vec3 path_trace_reinforcement_iterative(hiprandState* d_rand_state, Ray ray, Surface* surfaces, AreaLight* light_planes, int light_plane_count, int surfaces_count){

    vec3 throughput = vec3(1.f);
    
    // RadianceVolume* current_radiance_volume;
    // int current_sector_x = -1;
    // int current_sector_y = -1;

    for (int i = 0; i < MAX_RAY_BOUNCES; i++){

        // Trace the path of the ray to find the closest intersection
        ray.closest_intersection(surfaces, light_planes, light_plane_count, surfaces_count);

        // We cannot update Q on the first bounce as it is the camera position,
        // not a point in the scene
        // if (i > 0){
        //     // Update Q
        //     // where x = ray.start, y = intersection.position
        //     // Check that a radiance volume has been found to update its sector
        //     if (current_radiance_volume && current_sector_x != -1 && current_sector_y != -1){
        //         radiance_map.temporal_difference_update_radiance_volume_sector(current_radiance_volume, current_sector_x, current_sector_y, ray.intersection, surfaces, light_planes);
        //         current_sector_x = -1;
        //         current_sector_y = -1;
        //     } 
        // }

        // Check what they ray intersected with...
        switch(ray.intersection.intersection_type){
            // Interescted with nothing, so no radiance
            case NOTHING:
                return vec3(0);
                break;
            
            // Intersected with light plane, so return its diffuse_p
            case AREA_LIGHT:
                return throughput * light_planes[ray.intersection.index].diffuse_p;
                break;

            // Intersected with a surface (diffuse)
            case SURFACE:

                // vec4 sampled_direction = vec4(0.f);
                // current_radiance_volume = radiance_map.importance_sample_ray_direction(ray.intersection, current_sector_x, current_sector_y, sampled_direction);
                // TODO: Remove for reinforcment learning
                float cos_theta;
                vec4 sampled_direction = sample_random_direction_around_intersection(d_rand_state, ray.intersection, cos_theta);

                vec3 BRDF = surfaces[ray.intersection.index].material.diffuse_c / (float)M_PI;
                // float cos_theta = dot(vec3(surfaces[ray.intersection.index].normal), vec3(sampled_direction));
                float rho = (1.f / (2.f * (float)M_PI));

                throughput *= (BRDF * cos_theta) / rho;
                
                vec4 start = ray.intersection.position + sampled_direction * 0.00001f;
                start.w = 1.f;
                ray = Ray(start, sampled_direction);
                break;
            }
    }
    return vec3(0);
}